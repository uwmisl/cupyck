#include "hip/hip_runtime.h"
/*
  sumexp.c is part of the NUPACK software suite
  Copyright (c) 2007 Caltech. All rights reserved.
  Coded by: Robert Dirks 7/2006, Justin Bois 1/2007

  This file contains the functions that calculate the sum of exponentials
  in order to calculate a given partition function matrix. 

  See pfuncUtilsHeader.h for more specific descriptions of each function.
*/

#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<float.h>
#include<string.h>

#include "pfuncUtilsHeader.h" //contains functions and structures
/* ******************************************* */
DEV
DBL_TYPE ExplHairpin( int i, int j, int seq[], int seqlength, int **etaN,
    energy_model_t *em) {
  //this version disallows nicks here

  DBL_TYPE energy = 0;
  int index;
  int nNicks = 0;

  index = EtaNIndex(i+0.5, j-0.5, seqlength);
  nNicks = etaN[ index][0];

  if( nNicks >= 1) return 0;

  if( nNicks == 0 && j-i <= 3) {
    return 0;
  }

  if( nNicks == 0) {
    energy = HairpinEnergy( i, j, seq, em);
  }

  if( energy == NAD_INFINITY) {
    return 0.0;
  }

  return EXP_FUNC( -energy/( kB*em->temp_k) );
}

/* ********************* */
DEV
DBL_TYPE SumExpMultiloops( int i, int j, int seq[], 
                          DBL_TYPE *Qms, DBL_TYPE *Qm, int seqlength,
                          int **etaN, energy_model_t *em){
  // Decomposes the region inside pair i,j into multiloops, i.e.
  // and excludes the possibility of "top level" nicks

  DBL_TYPE sum_exp = 0.0;
  DBL_TYPE bp_penalty = 0.0;
  DBL_TYPE extraTerms;

  int d; // d is the left base of a rightmost paired base between i, j.
 
  if( ( seq[i]) + ( seq[j]) == 5) {
    for( d = i+3; d <= j - 2; d++) {
      //reset loop parameters
      bp_penalty = 0.0;
      if( etaN[EtaNIndex_same( d-0.5, seqlength)][0] == 0 ) {
        
        if( seq[i] != BASE_C  && seq[j] != BASE_C) {
          bp_penalty += em->at_penalty;
        }

        extraTerms = EXP_FUNC( -( em->alpha_1 + em->alpha_2 + bp_penalty) 
                          / (kB*em->temp_k) );
        if( em->dnarnacount == COUNT) 
          extraTerms = 1;

        sum_exp += Qm[ pf_index( i+1, d-1, seqlength)] *
          Qms[ pf_index(d, j-1, seqlength)] * extraTerms;
      }
      
    }
  }

  return sum_exp;  
}

/* *********************************************** */
DEV
DBL_TYPE SumExpExteriorLoop( int i,int j, int seq[], int seqlength, 
                            DBL_TYPE *Q, int *nicks, int **etaN, energy_model_t
                            *em) {

  DBL_TYPE sumExp = 0.0;
  DBL_TYPE bp_penalty = 0.0;
  int multiNick = -1;
  int index_ij;
  int leftIndex;
  int nNicks;
  int n;
  int iNicked, jNicked;
  DBL_TYPE extraTerms;

  index_ij = EtaNIndex(i+0.5, j-0.5, seqlength);
  iNicked = jNicked = FALSE;

  if( etaN[ EtaNIndex_same(j-0.5, seqlength)][0] != 0) {
    jNicked = TRUE;
  }

  if( etaN[ EtaNIndex_same(i+0.5, seqlength)][0] != 0) {
    iNicked = TRUE;
  }

  if( ( seq[i]) + ( seq[j]) == 5) {
    bp_penalty = 0.0;

    if( seq[i] != BASE_C  && seq[j] != BASE_C) {
      bp_penalty = em->at_penalty;
    }

    nNicks = etaN[ index_ij][0];
    leftIndex = etaN[ index_ij ][1];

    //treat each nick as rightmost nick in paired interval
    //All non-nicked cases handled outside this loop
    for( n = 0; n <= nNicks-1; n++) {
      multiNick = nicks[ leftIndex + n];

      extraTerms = 
        EXP_FUNC( -1*(bp_penalty)/(kB*em->temp_k));

      if( em->dnarnacount == COUNT) 
        if( extraTerms != 0) extraTerms = 1;

      if( (iNicked == FALSE && jNicked == FALSE) ||
         (i == j - 1) || 
         (multiNick == i && jNicked == FALSE) ||
         (multiNick == j-1 && iNicked == FALSE ) ) {

           sumExp +=
             Q[ pf_index(i+1, multiNick, seqlength)]*
             Q[ pf_index( multiNick+1, j-1, seqlength)] *
             extraTerms;
      }
    }
  }
  return sumExp;  
}

/* *********************************************** */
DEV
void fastILoops( int i, int j, int L, int seqlength, int seq[],
                 int **etaN, DBL_TYPE *Qb, DBL_TYPE *Qx, DBL_TYPE *Qx_2,
                 energy_model_t *em) {

  int size;
  int pf_ij = pf_index( i, j, seqlength);
  DBL_TYPE extraTerms;

  int isEndNicked = FALSE;
  if( etaN[ EtaNIndex( i-0.5,i-0.5, seqlength)][0] == 1 || 
     etaN[ EtaNIndex( j+0.5,j+0.5, seqlength)][0] == 1) 
    isEndNicked = TRUE;
  if( L >= 12) {
    makeNewQx( i, j, seq, seqlength, etaN, Qb, Qx, em);
  }

  //Use extensible cases              
  if( CanPair( seq[ i], seq[j]) == TRUE) {
    for( size = 8; size <= L - 4; size++) {
      
      extraTerms = EXP_FUNC( -InteriorMM( seq[i], seq[j], seq[i+1], 
                                     seq[j-1], em)/(kB*em->temp_k));
      if( em->dnarnacount == COUNT) 
        extraTerms = 1;
      
      Qb[ pf_ij] += 
        Qx[ fbixIndex( j-i, i, size, seqlength)] * 
        extraTerms;
    }
  }

  if( L >= 12 && i != 0 && j != seqlength -1 && isEndNicked == FALSE) {
    extendOldQx( i, j, seqlength, Qx,Qx_2, em );
  }
  
  /* Add in inextensible cases */  
  if( CanPair( seq[ i], seq[j]) == TRUE) {
    //first check inextensible cases

    Qb[ pf_ij] += SumExpInextensibleIL( i,j, seq, seqlength, Qb,  etaN, em);

  } 
}

/* *************** */

/* Qs, Qms  Recursion */
DEV
void MakeQs_Qms( int i, int j, int seq[], int seqlength, 
                DBL_TYPE *Qs, DBL_TYPE *Qms, DBL_TYPE *Qb,
                int *nicks, int **etaN, energy_model_t *em) {

  int d; //base pair is i,d
  DBL_TYPE bp_penalty = 0.0;
  int pf_ij = pf_index( i, j, seqlength);

  DBL_TYPE extraTerms;
  int nNicks;
  int index_ij = EtaNIndex( i+0.5, j-0.5, seqlength);
  int start;

  nNicks = etaN[ index_ij][0];
  if( nNicks >= 1) {
    start = nicks[ etaN[ index_ij][1] + nNicks - 1]+1;
  }
  else {
    start = i+4;
  }

  for( d = start; d <= j; d++) {
    bp_penalty = 0.0;
    
    if( CanPair( seq[i], seq[ d]) == TRUE &&
       ( seq[i]) + ( seq[d]) == 5) {
         
         if( seq[i] != BASE_C && seq[d] != BASE_C) {
           bp_penalty = em->at_penalty;
         }

         extraTerms = EXP_FUNC( -(NickDangle( d+1,j,nicks, etaN,
                              FALSE, seq,seqlength, em) + 
                              bp_penalty)/(kB*em->temp_k) );

         if( em->dnarnacount == COUNT) 
           extraTerms = 1;

         Qs[ pf_ij] += Qb[ pf_index( i, d, seqlength) ] * 
           extraTerms;

         // ******************** 

         extraTerms =  ExplDangle( d+1, j, seq, seqlength, em) * 
           EXP_FUNC( -(bp_penalty + em->alpha_2 + 
                 em->alpha_3*(j-d))/(kB*em->temp_k) );

         if( em->dnarnacount == COUNT) 
           extraTerms = 1;
         Qms[ pf_ij] += Qb[ pf_index( i, d, seqlength) ] * 
           extraTerms;
    }
  }
}


/* ******************************* */
/* Q, Qm Recursions */
DEV
void MakeQ_Qm_N3( int i, int j, int seq[], int seqlength, 
                 DBL_TYPE *Q, DBL_TYPE *Qs, 
                 DBL_TYPE *Qms, DBL_TYPE *Qm,
                 int *nicks, int **etaN, energy_model_t *em) {
  int d;//left base of rightmost base pair.
  int pf_ij = pf_index( i, j, seqlength);

  DBL_TYPE extraTerms;
  Q[ pf_ij] = NickedEmptyQ( i, j, nicks, seq, seqlength, etaN, em);

  for( d = i; d <= j - 1; d++) {
    if( etaN[ EtaNIndex_same(d-0.5, seqlength)][0] == 0 || d == i ) {

      Q[ pf_ij] += Q[ pf_index(i, d-1, seqlength)] *
        Qs[ pf_index( d, j, seqlength)];

      if( em->dnarnacount == COUNT) 
        extraTerms = 1;
      else 
        extraTerms = ExplDangle( i, d-1, seq, seqlength, em) *
          EXP_FUNC( -(em->alpha_3)*(d-i)/(kB*em->temp_k));

      if( etaN[ EtaNIndex_same( d-0.5, seqlength)][0] == 0) { 
        //otherwise Qm not possible
       
        if( etaN[ EtaNIndex(i+0.5, d-0.5, seqlength)][0] == 0 ) {
          Qm[ pf_ij] += Qms[ pf_index( d, j, seqlength)] *
           extraTerms; //Single Pair
        }

        if( d >= i+2) {
         Qm[ pf_ij]+= Qm[ pf_index( i, d - 1, seqlength) ] *
           Qms[ pf_index( d, j, seqlength) ];
        }
      }
    }
  }
}

/* ******************************************* */

/* Functions in Q recursion */
// must be calculated after Qb, Qpk of same length

DEV void makeNewQx( int i, int j, int seq[], int seqlength, 
               int **etaN, DBL_TYPE Qb[], DBL_TYPE Qx[], energy_model_t *em) {
                 
  /*Determine the new entries of Qx(i,j,size) that are not extended 
  versions of Qx(i+1, j-1, size-2) */

  DBL_TYPE energy;
  int d, e; //Internal pair.(d, e will be restricted to special cases)

  int size, L1, L2; //size parameters: L1 + L2 = size, L1 = h-i-1, L2 = j-m-1

  //Add in all the cases that are not an extended version of an
  //extensible case.

  //Case 1:  L1 = 4, L2 >= 4;
  L1 = 4;
  d = i + L1 + 1;

  for( L2 = 4; L2 <= j - d - 2; L2++) {
    size = L1 + L2;
    e = j - L2 - 1;

    if( CanPair( seq[d], seq[e]) == TRUE &&
      (etaN[ EtaNIndex(i+0.5, d-0.5,seqlength)][0] == 0) &&
      (etaN[ EtaNIndex(e+0.5, j-0.5,seqlength)][0] == 0) ) {

        energy = asymmetryEfn( L1, L2, size, em) + InteriorMM( seq[e], seq[d],
            seq[e+1], seq[d-1], em);
        /*Exclude the i-j stacking energy here, just in case i-j 
        don't pair */

        if( em->dnarnacount == COUNT) 
          energy = 0;

        Qx[ fbixIndex( j-i, i, size, seqlength) ] += 
          EXP_FUNC(-energy/(kB*em->temp_k))*Qb[ pf_index(d, e, seqlength)];
    }
  }

  //Case 2  L1 > 4, L2 = 4
  L2 = 4;
  e = j - L2 -1;
  for( L1 = 5; L1 <= e-i-2; L1++) {
    size = L1 + L2;
    d = i + L1 + 1;

    if( CanPair( seq[d], seq[e]) == TRUE &&
      (etaN[ EtaNIndex(i+0.5, d-0.5,seqlength)][0] == 0) &&
      (etaN[ EtaNIndex(e+0.5, j-0.5,seqlength)][0] == 0) ) {

        energy = asymmetryEfn( L1, L2, size, em) + InteriorMM( seq[e], seq[d],
            seq[e+1], seq[d-1], em);
        /*Exclude the i-j stacking energy here, just in case i-j 
        don't pair */

        if( em->dnarnacount == COUNT) 
          energy = 0.0;

        Qx[ fbixIndex( j-i, i, size, seqlength)] +=
          EXP_FUNC(-energy/(kB*em->temp_k))*Qb[ pf_index(d, e, seqlength)];
    }
  }
}

/* ************************** */
DEV
void extendOldQx( int i, int j, int seqlength, DBL_TYPE Qx[], DBL_TYPE Qx_2[],
    energy_model_t *em) {
  /* Extends all entries of Qx */
  
  int size;
  DBL_TYPE oldSizeEnergy;
  DBL_TYPE newSizeEnergy;
  
  for( size = 8; size <= (j - i + 1) - 4; size++) {
    if( size <= 30) {
      oldSizeEnergy = em->loop37[ size - 1];
    }
    else {
      oldSizeEnergy = em->loop37[ 30 - 1];
      oldSizeEnergy += sizeLog(size, em);  //1.75*kB*TEMP_K*log( size/30.0); 
    }

    if( size + 2 <= 30) {
      newSizeEnergy = em->loop37[ size+2 - 1];
    }
    else {
      newSizeEnergy = em->loop37[ 30 - 1];
      newSizeEnergy += sizeLog (size+2, em); //1.75*kB*TEMP_K*log( (size+2)/30.0); 
    }

    if( em->dnarnacount == COUNT) 
      newSizeEnergy = oldSizeEnergy;
    
    Qx_2[ fbixIndex( j-i+2, i-1, size+2, seqlength)] = 
      Qx[ fbixIndex( j-i, i, size, seqlength)] * 
      EXP_FUNC( -(newSizeEnergy - oldSizeEnergy)/(kB*em->temp_k));
  }
}


/* ************************ */
DEV
DBL_TYPE SumExpInextensibleIL( int i, int j, int seq[], int seqlength, 
                              DBL_TYPE Qb[], int **etaN, energy_model_t *em) {
  /* This finds the minimum energy IL that has a special energy 
  calculation, i.e. small loops, bulge loops or GAIL case.  None of 
  these are allowed to be nicked
  */

  DBL_TYPE energy;
  DBL_TYPE sumexp = 0.0;
  int d, e; //Internal pair.(h, m will be restricted to special cases)  
  int L1, L2; //size parameters: L1 + L2 = size, L1 = h-i-1, L2 = j-m-1

  /* Consider "small" loops with special energy functions */

  for( L1 = 0; L1 <= 3; L1++) {
    d = i + L1 + 1;
    for( L2 = 0; L2 <= MIN( 3, j-d-2); L2++) {
      e = j - L2 - 1;

      if( CanPair( seq[d], seq[e]) == TRUE &&
         (etaN[ EtaNIndex(i+0.5, d-0.5,seqlength)][0] == 0) &&
         (etaN[ EtaNIndex(e+0.5, j-0.5,seqlength)][0] == 0) ) {

           energy = InteriorEnergy( i, j, d, e, seq, em);

           sumexp += EXP_FUNC( -energy/(kB*em->temp_k)) *
             Qb[ pf_index( d, e, seqlength)];
      }
    }
  }

  /* Next consider large bulges or large asymmetric loops */
  // Case 2a  L1 = 0,1,2,3, L2 >= 4;
  for( L1 = 0; L1 <= 3; L1++) {
    d = i + L1 + 1;
    for( L2 = 4; L2 <= j - d - 2; L2++) {
      e = j - L2 - 1;

      if( CanPair( seq[d], seq[e]) == TRUE &&
         (etaN[ EtaNIndex(i+0.5, d-0.5,seqlength)][0] == 0) &&
         (etaN[ EtaNIndex(e+0.5, j-0.5,seqlength)][0] == 0) ) {

           energy = InteriorEnergy( i, j, d, e, seq, em);

           sumexp += EXP_FUNC( -energy/(kB*em->temp_k)) *
             Qb[ pf_index( d, e, seqlength)]; 
      }
    }
  }

  // Case 2b L1 >= 4, L2 = 0,1,2,3;
  for( L2 = 0; L2 <= 3; L2++) {
    e = j - L2 - 1;
    for( L1 = 4;  L1 <= e - i - 2; L1++) {
      d = i + L1 + 1;

      if( CanPair( seq[d], seq[e]) == TRUE &&
         (etaN[ EtaNIndex(i+0.5, d-0.5,seqlength)][0] == 0) &&
         (etaN[ EtaNIndex(e+0.5, j-0.5,seqlength)][0] == 0) ) {

           energy = InteriorEnergy( i, j, d, e, seq, em);

           sumexp += EXP_FUNC( -energy/(kB*em->temp_k)) *
             Qb[ pf_index( d, e, seqlength)];
      }
    }
  }

  return sumexp;
}




