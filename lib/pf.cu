#include "hip/hip_runtime.h"
/*
  pf.c is part of the NUPACK software suite
  Copyright (c) 2007 Caltech. All rights reserved.
  Coded by: Robert Dirks 3/2006, Justin Bois 1/2007
            Asif Khan 8/2009 Brian Wolfe 10/2009
  This file moves the partition function algorithm to a function, so
  that it can be more readily used as in a library.  In addition,
  scaling is incorporated to allow for longer sequences.
*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <assert.h>

#include "pfuncUtilsHeader.h" //contains functions and structures
#include "DNAExternals.h"


/* ************************************************ */
// This is the main function for computing partition functions.

DEV
int getNodesFirstEntry(int L, int rank, int seqLen, int N) {
    int split, rem, last;
      last = seqLen - L + 1;
        if (rank >= N || rank < 0)
              return last; 
          split = (int)(last/N);
            rem = last % N;
              return rank*split + MIN(rank, rem);
}

DEV
int IsProcessorInUse(int rank, int L, int seqLen, int numCPUs ) {
    return (getNodesFirstEntry(L,rank,seqLen,numCPUs) != seqLen - L + 1);
}

__constant__ energy_model_t *energies;
__constant__ DBL_TYPE t_lo;
__constant__ DBL_TYPE t_hi;
__constant__ DBL_TYPE t_step;
DEV
energy_model_t* get_energy_model(DBL_TYPE temp_k) {
  assert(t_lo <= temp_k && temp_k <= t_hi);
  int index = round((temp_k - t_lo) / t_step);
  return energies + index;
}

#define cudaCheck(call) {\
  hipError_t e = (call); \
  if(e != hipSuccess) {\
    fprintf(stderr, #call ": %s", hipGetErrorString(e)); \
    abort(); \
  }\
}\


class PFMemory {
  public:
    DBL_TYPE *Q;
    DBL_TYPE *Qb;
    DBL_TYPE *Qm;

    DBL_TYPE *Qs;
    DBL_TYPE *Qms;

    DBL_TYPE *Qx;
    DBL_TYPE *Qx_1;
    DBL_TYPE *Qx_2;

    int *seq;
    int **etaN;
    int *etaN_space;

    int arraySize;
    int seqlength;

    PFMemory() {}
    ~PFMemory() {}
    void init(int);
    void free();
    __device__ void clear();
};
void PFMemory::init(int seqlength) {
  this->seqlength = seqlength;
  arraySize = seqlength*(seqlength+1)/2+(seqlength+1);

  cudaCheck(hipMalloc(&Q, arraySize * sizeof(DBL_TYPE)));
  cudaCheck(hipMalloc(&Qb, arraySize * sizeof(DBL_TYPE)));
  cudaCheck(hipMalloc(&Qm, arraySize * sizeof(DBL_TYPE)));

  cudaCheck(hipMalloc(&Qs, arraySize * sizeof(DBL_TYPE)));
  cudaCheck(hipMalloc(&Qms, arraySize * sizeof(DBL_TYPE)));

  cudaCheck(hipMalloc(&Qx, arraySize/2 * sizeof(DBL_TYPE)));
  cudaCheck(hipMalloc(&Qx_1, arraySize/2 * sizeof(DBL_TYPE)));
  cudaCheck(hipMalloc(&Qx_2, arraySize/2 * sizeof(DBL_TYPE)));

  cudaCheck(hipMalloc(&seq, (seqlength + 1) * sizeof(int)));
  cudaCheck(hipMalloc(&etaN, arraySize * sizeof(int*)));
  cudaCheck(hipMalloc(&etaN_space, arraySize * 2 * sizeof(int)));

}
void PFMemory::free() {
  cudaCheck(hipFree(Q));
  cudaCheck(hipFree(Qb));
  cudaCheck(hipFree(Qm));

  cudaCheck(hipFree(Qs));
  cudaCheck(hipFree(Qms));

  cudaCheck(hipFree(Qx));
  cudaCheck(hipFree(Qx_1));
  cudaCheck(hipFree(Qx_2));

  cudaCheck(hipFree(seq));
  cudaCheck(hipFree(etaN));
  cudaCheck(hipFree(etaN_space));
}

__device__ void PFMemory::clear() {
  memset(Q, 0, arraySize * sizeof(DBL_TYPE));
  memset(Qb, 0, arraySize * sizeof(DBL_TYPE));
  memset(Qm, 0, arraySize * sizeof(DBL_TYPE));

  memset(Qs, 0, arraySize * sizeof(DBL_TYPE));
  memset(Qms, 0, arraySize * sizeof(DBL_TYPE));

  memset(Qx, 0, arraySize/2 * sizeof(DBL_TYPE));
  memset(Qx_1, 0, arraySize/2 * sizeof(DBL_TYPE));
  memset(Qx_2, 0, arraySize/2 * sizeof(DBL_TYPE));

  memset(seq, 0, (seqlength + 1) * sizeof(int));
  memset(etaN, 0, arraySize * sizeof(int*));
  memset(etaN_space, 0, arraySize * 2 * sizeof(int));
}

__constant__ PFMemory *pf_mem;

GLB
void pfuncFullWithSymHelper(DBL_TYPE *pf, int ** inputSeqs, int * seqlengths,
    int * nStrands_arr, int * permSymmetries, DBL_TYPE * temps) {

  int * inputSeq = inputSeqs[blockIdx.x];
  int seqlength = seqlengths[blockIdx.x];
  int nStrands = nStrands_arr[blockIdx.x];
  int permSymmetry = permSymmetries[blockIdx.x];
  energy_model_t *em = get_energy_model(temps[blockIdx.x]);

  //complexity: 3 = N^3, 4 = N^4, 5 = N^5, 8 = N^8
  //naType: DNA = 0, RNA = 1
  //dangles: 0 = none, 1 = normal, 2 = add both

  __shared__ int *seq;

  __shared__ DBL_TYPE *Q;
  __shared__ DBL_TYPE *Qb;
  __shared__ DBL_TYPE *Qm; //O(N^2)


  //N^3 arrays
  __shared__ DBL_TYPE *Qx;
  __shared__ DBL_TYPE *Qx_1;
  __shared__ DBL_TYPE *Qx_2;
  __shared__ DBL_TYPE *Qs;
  __shared__ DBL_TYPE *Qms;

  /*
  The above matrices are dynamically allocated matrices that
  contain partition functions restricted to a subsequence of the
  strand.  Each of the above should be accessed by the call
  to Q[ pf_index(i, j)] indicate the partition function between
  i and j, inclusive. 

  They are described in the paper mentioned above.
  */

  int i, j; // the beginning and end bases for Q;
  int L; //This the length of the current subsequence 
  int pf_ij; //index for O(N^2) matrixes; used to reduce calls to pf_index;
  DBL_TYPE returnValue;


  int iMin;
  int iMax;
  

  __shared__ int nicks[ MAXSTRANDS];  //the entries must be strictly increasing
  if (threadIdx.x == 0) {
    for (i=0;i<MAXSTRANDS;i++){
      nicks[i]=-1;
    }
  }
  //nicks[i] = N means a strand ends with base N, and a new one starts at N+1
  // isNicked[n] is 0 if no nick at n, 1 otherwise

  __shared__ int **etaN;
  __shared__ int *etaN_space;

  if (threadIdx.x == 0) {
    PFMemory *mem = pf_mem + blockIdx.x;
    mem->clear();
    Q    = mem->Q;
    Qb   = mem->Qb;
    Qm   = mem->Qm;
    Qx   = mem->Qx;
    Qx_1 = mem->Qx_1;
    Qx_2 = mem->Qx_2;
    Qs   = mem->Qs;
    Qms  = mem->Qms;
    seq  = mem->seq;
    etaN = mem->etaN;
    etaN_space = mem->etaN_space;

    processMultiSequence( inputSeq, seqlength, nStrands, seq, nicks);

    // Allocate and Initialize Matrices
    int arraySize = seqlength*(seqlength+1)/2+(seqlength+1);

    for (int i = 0; i < arraySize; ++i) {
      etaN[i] = etaN_space + (2 * i);
    }
    InitEtaN( etaN, nicks, seqlength);
    nonZeroInit( Q, seq, seqlength, em);

  }

  for( L = 1; L <= seqlength; L++) {
    /* Calculate all sub partition functions for
    distance = 0, then 1, then 2.... */

    if (threadIdx.x == 0) {
      manageQx( &Qx, &Qx_1, &Qx_2, L-1, seqlength);   
    }
    __syncthreads();

    iMin = getNodesFirstEntry(L, threadIdx.x, seqlength, blockDim.x);
    iMax = getNodesFirstEntry(L, threadIdx.x + 1, seqlength, blockDim.x) - 1;
    /*
    //Default without parallelization
    iMin = 0;
    iMax = seqlength - L; 
    */

  
    int active = IsProcessorInUse(threadIdx.x, L, seqlength, blockDim.x);
    for( i = iMin; i <= iMax && active; i++) {
      j = i + L - 1;
      pf_ij = pf_index( i, j, seqlength);
      /* Recursions for Qb.  See figure 13 of paper */
      /* bp = base pairs, pk = pseudoknots */
      if( CanPair( seq[ i], seq[ j]) == FALSE) {
        Qb[ pf_ij] = 0.0; //scaling still gives 0
      }
      else {
        Qb[ pf_ij] = ExplHairpin( i, j, seq, seqlength, etaN, em);

        //no nicked haripins allowed in previous function
        if( etaN[ EtaNIndex(i+0.5, i+0.5, seqlength)][0] == 0 &&
           etaN[ EtaNIndex(j-0.5, j-0.5, seqlength)][0] == 0) {
             //regular multiloop.  No top-level nicks
             
              Qb[ pf_ij] += SumExpMultiloops(i, j, seq, Qms, Qm,
                                            seqlength, etaN, em);
        }

        if( etaN[ EtaNIndex(i+0.5, j-0.5, seqlength)][0] >= 1) {
          //Exterior loop (created by nick)
          Qb[ pf_ij] += SumExpExteriorLoop( i, j, seq, seqlength, 
                                           Q, nicks, etaN, em); 
        }

      }

      fastILoops( i, j, L, seqlength, seq, etaN, Qb, Qx, Qx_2, em);


      /* Recursions for Qms, Qs */
      MakeQs_Qms( i, j, seq, seqlength, Qs, Qms, Qb, nicks, etaN, em);
      
      /* Recursions for Q, Qm, Qz */
      MakeQ_Qm_N3( i, j, seq, seqlength, Q, Qs, Qms, Qm,
                  nicks,etaN, em);


    }
    __syncthreads();
  }

  //adjust this for nStrands, symmetry at rank == 0 node
  if (threadIdx.x == 0) {
    returnValue = EXP_FUNC(
        -1*(em->bimolecular + em->salt_correction)*(nStrands-1)/
        (kB*em->temp_k)
      ) * Q[ pf_index(0,seqlength-1, seqlength)]/((DBL_TYPE) permSymmetry);


    pf[blockIdx.x] = returnValue;
  }
}
/* ****** */

DBL_TYPE * pf;
int ** intSeqs;
int * seqlengths;
int * nStrands_arr;
int * permSymmetries;
DBL_TYPE * temps;
int nblocks;
int nthreads;

energy_model_t *em_host;
energy_model_t *em_dev;

PFMemory *pfm_host;
PFMemory *pfm_dev;

extern "C" void pfuncInitialize(
    int nblocks_in,
    int nthreads_in,
    int max_seqlen,
    DBL_TYPE temp_lo, DBL_TYPE temp_hi, DBL_TYPE temp_step,
    DBL_TYPE sodium_conc, DBL_TYPE magnesium_conc,
    int long_helix, int dangletype, int dnarnacount) {

  nblocks = nblocks_in;
  nthreads = nthreads_in;
  SODIUM_CONC = sodium_conc;
  MAGNESIUM_CONC = magnesium_conc;
  USE_LONG_HELIX_FOR_SALT_CORRECTION = long_helix;
  DANGLETYPE = dangletype;
  DNARNACOUNT = dnarnacount;

  // Load Energy Models
  int ntemps = ceil((temp_hi - temp_lo) / temp_step) + 1;
  temp_lo += ZERO_C_IN_KELVIN;
  temp_hi += ZERO_C_IN_KELVIN;

  em_host = (energy_model_t*)malloc(ntemps * sizeof(energy_model_t));

  DBL_TYPE temp_k = temp_lo;
  for(int i = 0; i < ntemps; ++i) {
    LoadEnergies(em_host + i, temp_k);
    temp_k += temp_step;
  }
  // Allocate device memory and copy
  hipMalloc(&em_dev, ntemps * sizeof(energy_model_t));
  hipMemcpy(em_dev, em_host, ntemps * sizeof(energy_model_t), hipMemcpyHostToDevice);

  // Set device constants
  cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(energies), &em_dev, sizeof(energy_model_t*)));
  cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(t_hi), &temp_hi, sizeof(DBL_TYPE)));
  cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(t_lo), &temp_lo, sizeof(DBL_TYPE)));
  cudaCheck(hipMemcpyToSymbol(HIP_SYMBOL(t_step), &temp_step, sizeof(DBL_TYPE)));

  cudaCheck(
    hipMallocManaged(&pf, nblocks * sizeof(DBL_TYPE))
  );
  cudaCheck(
    hipMallocManaged(&seqlengths, nblocks * sizeof(int))
  );
  cudaCheck(
    hipMallocManaged(&nStrands_arr, nblocks * sizeof(int))
  );
  cudaCheck(
    hipMallocManaged(&permSymmetries, nblocks * sizeof(int))
  );
  cudaCheck(
    hipMallocManaged(&temps, nblocks * sizeof(DBL_TYPE))
  );
  cudaCheck(
    hipMallocManaged(&intSeqs, nblocks * sizeof(int*))
  );

  for(int i = 0; i < nblocks; ++i) {
    cudaCheck(
      hipMallocManaged(&(intSeqs[i]), (MAXSEQLENGTH + 1) * sizeof(int))
    );
  }

  // perform allocations
  pfm_host = new PFMemory[nblocks];
  for(int i = 0; i < nblocks; ++i) { pfm_host[i].init(max_seqlen); }

  hipMalloc(&pfm_dev, nblocks * sizeof(PFMemory));
  hipMemcpy(pfm_dev, pfm_host, nblocks * sizeof(PFMemory), hipMemcpyHostToDevice);

  // copy symbol
  hipMemcpyToSymbol(HIP_SYMBOL(pf_mem), &pfm_dev, sizeof(PFMemory*));
}

extern "C" void pfuncShutdown() {
  // free device memory for energy models
  cudaCheck(hipFree(em_dev));
  // free host memory for energy models
  free(em_host);

  // free unified memory for parameters
  cudaCheck(hipFree(pf));
  cudaCheck(hipFree(seqlengths));
  cudaCheck(hipFree(nStrands_arr));
  cudaCheck(hipFree(permSymmetries));
  cudaCheck(hipFree(temps));
  for (int i = 0; i < nblocks; ++i) {
    cudaCheck(hipFree(intSeqs[i]));
  }
  cudaCheck(hipFree(intSeqs));

  // free device memory for PF arrays
  for(int i = 0; i < nblocks; ++i) {
    pfm_host[i].free();
  }
  // free device memory for PF array pointers
  cudaCheck(hipFree(pfm_dev));
  // free host memory for PF array pointers
  delete [] pfm_host;

}

extern "C" void pfuncMulti(char ** inputSeqs, int nseqs, int * permSym,
    DBL_TYPE * temp_Cs, DBL_TYPE * result) {

  for (int s = 0; s < nseqs; s += nblocks) {
    int njobs = MIN(nblocks, nseqs - s);
    for(int i = 0; i < njobs; ++i) {
      int len = strlen(inputSeqs[s + i]);
      convertSeq(inputSeqs[s + i], intSeqs[i], len);
      seqlengths[i] = getSequenceLengthInt(intSeqs[i], nStrands_arr + i);
      permSymmetries[i] = permSym[s + i];
      temps[i] = temp_Cs[s + i] + ZERO_C_IN_KELVIN;
    }

    pfuncFullWithSymHelper<<<njobs, nthreads>>>(
        pf, intSeqs, seqlengths, nStrands_arr, permSymmetries, temps
    );
    hipDeviceSynchronize();

    for(int i = 0; i < njobs; ++i) {
      result[s + i] = pf[i];
    }
  }
}
