#include "hip/hip_runtime.h"
/*
  init.c is part of the NUPACK software suite
  Copyright (c) 2007 Caltech. All rights reserved.
  Coded by: Robert Dirks 3/2006, Justin Bois 1/2007

  Functions to be run once at the beginning of the
  partition function algorithm
*/
#include<stdio.h>
#include<math.h>
#include<stdlib.h>
#include<string.h>
#include<ctype.h>
#include<assert.h>

#include "pfuncUtilsHeader.h"
#include "DNAExternals.h"

/* **************************** */
int getSequenceLength( char *seq, int *nStrands /*, seq2, nicks*/) {
  
  int i; //position in sequence
  int done = FALSE;
  int seqlength = 0;
  char tmpC;

  i = 0;
  *nStrands = 1;
  while( done == FALSE) {
    tmpC = toupper( seq[i]);
    if( tmpC == '+') {
      (*nStrands)++;
      //nicks[ seqlength] = 1;
    }
    else if( tmpC != 'A' && tmpC != 'T' && tmpC != 'C'
            && tmpC != 'G' && tmpC != 'U') {
              done = TRUE; 
            }
    else {
      //seq2[ seqlength] = tmpC;
      seqlength++;
    }
    i++;
  }

  if( seqlength > MAXSEQLENGTH) {
    printf("Sequences longer than maximum of %d\n", MAXSEQLENGTH);
    assert(0);
  }

  return seqlength;
}

/* **************************** */
int getSequenceLengthInt( int seq[], int *nStrands /*, seq2, nicks*/) {

  int i; //position in sequence
  int done = FALSE;
  int seqlength = 0;
  int tmpC;

  i = 0;
  *nStrands = 1;
  while( done == FALSE) {
    tmpC = seq[i];
    if( tmpC == STRAND_PLUS) {
      (*nStrands)++;
      //nicks[ seqlength] = 1;
    }
    else if( tmpC != BASE_A && tmpC != BASE_T && tmpC != BASE_C
            && tmpC != BASE_G && tmpC != BASE_U) {
              done = TRUE; 
            }
    else {
      //seq2[ seqlength] = tmpC;
      seqlength++;
    }
    i++;
  }

  if( seqlength > MAXSEQLENGTH) {
    printf("Sequences longer than maximum of %d\n", MAXSEQLENGTH);
    assert(0);
  }

  return seqlength;
}

/* *********** */
DEV
void processMultiSequence( int inputSeq[], int seqlength, int nStrands,
                           int seq[], int nicks[]) {
  
  int i, j;
  int nNick = 0;
  int done;

  j = 0;
  for( i = 0; i < seqlength; i++) {
    done = FALSE;
    while( !done) {
      if( inputSeq[j] == BASE_A || inputSeq[j] == BASE_C || inputSeq[j] == BASE_G ||
        inputSeq[j] == BASE_T || inputSeq[j] == BASE_U) {
        done = TRUE;
        seq[i] = inputSeq[j];
      }
      else if( inputSeq[j] == STRAND_PLUS) {
        nicks[nNick++] = i-1;
      }
      j++;

      if( j >= seqlength + nStrands && !done) {
        printf("\nError in processing sequence:\n%d\n", inputSeq[0]);
        printf("seqlength = %d, nStrands = %d\n", seqlength, nStrands);
        assert(0);
      }
    }
  }
  seq[ seqlength] = -1;
}


/* ********************************************* */
DEV
void InitLDoublesMatrix( DBL_TYPE **Q, int size, char name[]) {
  // Allocate cleared memory for a DBL_TYPEs matrix.
  *Q =  (DBL_TYPE *) malloc( size * sizeof( DBL_TYPE));
  memset(*Q, 0, size * sizeof(DBL_TYPE));
  if( *Q == NULL) {
    printf("InitLDoublesMatrix: unable to allocate %lu bytes for %s!\n", size * sizeof( DBL_TYPE),  name);
    assert(0);
  }
}

void ClearLDoublesMatrix(DBL_TYPE **Q, int size, char name[]) {
  memset(*Q, 0,size * sizeof(DBL_TYPE));
}

/* ******************************************** */
DEV
void nonZeroInit( DBL_TYPE Q[], int seq[], int seqlength, energy_model_t *em) {
  // Set Q[i, i-1] = 1.
  int i;

  for( i = 0; i <= seqlength; i++) {
    Q[ pf_index(i, i-1, seqlength)] = ExplDangle(i,i-1,seq,seqlength, em);
  }
}


/* *************************************************** */
DEV
void manageQx( DBL_TYPE **Qx, DBL_TYPE **Qx_1, DBL_TYPE **Qx_2, int len, int seqlength) {
  // Allocate and deallocate QbIx matrices

  int i;
  DBL_TYPE *temp;
  int arraySize = seqlength*(seqlength+1)/2 + (seqlength+1);

  if( len > 11) {

    temp = *Qx;
    *Qx = *Qx_1;
    *Qx_1 = *Qx_2;
    *Qx_2 = temp;

    for(i = 0; i < arraySize/2; ++i) {
      (*Qx_2)[i] = 0;
    }

  }
}


/* ************************************** */
DBL_TYPE computeSaltCorrection(DBL_TYPE sodiumConc, DBL_TYPE magnesiumConc,
			       int useLongHelix, DBL_TYPE temp_k) {

  // No correction for RNA since we don't have parameters
  if (DNARNACOUNT != DNA || (sodiumConc == 1.0 && magnesiumConc == 0.0)) { 
    return 0.0;
  }

  // Ignore magnesium for long helix mode (not cited why, for consistency with Mfold)
  if (useLongHelix) { 
    return -(0.2 + 0.175*log(sodiumConc)) * temp_k / 310.15;
  }

  return -0.114*log(sodiumConc + 3.3*sqrt(magnesiumConc)) * temp_k / 310.15;
}

#define fgets(args...) assert(fgets(args) != 0)
/* ************************************** */
void LoadEnergies(energy_model_t *em, DBL_TYPE temp_k) {
  
  const char *default_param_files[] = { "dna1998", "rna1995", "rna1999"};
  
  DBL_TYPE H_loop37[90];  
  DBL_TYPE H_tloop_energy[4096];
  DBL_TYPE H_triloop_energy[2048];
  DBL_TYPE H_MMEnergiesHP[6*16];
  DBL_TYPE H_MMEnergiesIL[256];
  DBL_TYPE H_IL_SInt2[16*36];
  DBL_TYPE H_IL_SInt4[256*36];
  DBL_TYPE H_IL_AsInt1x2[64*36];
  DBL_TYPE H_dangle_energy[48];
  DBL_TYPE H_asymmetry_penalty[4];
  DBL_TYPE H_max_asymmetry;
  DBL_TYPE H_Stack[36];
  DBL_TYPE H_ALPHA_1, H_ALPHA_2, H_ALPHA_3, H_BETA_1, H_BETA_2, 
  H_BETA_3, H_BETA_1M, H_BETA_1P;
  DBL_TYPE H_POLYC3, H_POLYCINT, H_POLYCSLOPE;
  DBL_TYPE H_AT_PENALTY;
  DBL_TYPE H_BIMOLECULAR;
  
  DBL_TYPE G_loop37[90];  
  DBL_TYPE G_tloop_energy[4096];
  DBL_TYPE G_triloop_energy[2048];
  DBL_TYPE G_MMEnergiesHP[6*16];
  DBL_TYPE G_MMEnergiesIL[256];
  DBL_TYPE G_IL_SInt2[16*36];
  DBL_TYPE G_IL_SInt4[256*36];
  DBL_TYPE G_IL_AsInt1x2[64*36];
  DBL_TYPE G_dangle_energy[48];
  DBL_TYPE G_asymmetry_penalty[4];
  DBL_TYPE G_max_asymmetry;
  DBL_TYPE G_Stack[36];
  DBL_TYPE G_ALPHA_1, G_ALPHA_2, G_ALPHA_3, G_BETA_1, G_BETA_2,
  G_BETA_3, G_BETA_1M, G_BETA_1P;
  DBL_TYPE G_POLYC3, G_POLYCINT, G_POLYCSLOPE;
  DBL_TYPE G_AT_PENALTY;
  DBL_TYPE G_BIMOLECULAR;
  DBL_TYPE water_conc;


  //temporary storage of data
  int nRead;
  int array[MAXLINE];
  char *token;
  char tetraloop[6] = "\0";
  char triloop[5] = "\0";
  int indexL, tmpIndex, index4;
  
  FILE *fp;	
  char line[MAXLINE];
  int i, j, k;
  char fileG[300] = "\0";
  char fileH[300] = "\0";
  char *nupackhome = NULL;
  char fileNameRoot[300] = "\0";
  
  static char parameterFileName[300] = "\0";
  

  em->temp_k = temp_k;
  em->dnarnacount = DNARNACOUNT;
  em->dangletype = DANGLETYPE;

  if( DNARNACOUNT == COUNT) {
    setParametersToZero(em);
    return;
  }
  
  // Get density of water
  water_conc = (DBL_TYPE) WaterDensity(temp_k - ZERO_C_IN_KELVIN);

  // Compute the salt correction
  em->salt_correction = computeSaltCorrection(
      SODIUM_CONC, MAGNESIUM_CONC,
      USE_LONG_HELIX_FOR_SALT_CORRECTION, temp_k);
  
  
  //Parameter file input.  If a path was given as a command line parameter,
  //follow this path and return an error if the parameter files cannot be found.
  //Otherwise, check the local directory first, and if the files are not there,
  //check the NUPACKHOME/parameters directory.
  
  if( DNARNACOUNT < USE_SPECIFIED_PARAMETERS_FILE) {
    strcpy( fileNameRoot, default_param_files[ DNARNACOUNT]);
    strcpy( parameterFileName, ""); //Used to check if parameter files need to be reloaded.
  }
  else if( DNARNACOUNT == USE_SPECIFIED_PARAMETERS_FILE) {
    strcpy( fileNameRoot, PARAM_FILE);
    strcpy( parameterFileName, PARAM_FILE); //store this to check if parameter reload is needed.
  }
  
  //check first for .dG parameter file using current directory as home
  strcpy( fileG, fileNameRoot);
  strcat( fileG, ".dG");
  
  if( !fileExists( fileG) ) {
    //if files not found, use environment variable NUPACKHOME as root
    nupackhome = getenv("NUPACKHOME");
    if( nupackhome != NULL) {
      strcpy( fileG, nupackhome);
      strcat( fileG, "/parameters/");
      strcat( fileG, fileNameRoot);
      strcat( fileG, ".dG");
    }
    else {
      fprintf(stderr, "Unable to find %s.dG and NUPACKHOME environment variable is not set\n",
             fileNameRoot);
      exit(1);
    }
  }
  
  if( ! fileExists( fileG)) {
    fprintf(stderr, "Unable to find file %s.dG locally or in NUPACKHOME = %s/parameters\n", 
           fileNameRoot, 
           nupackhome);
    fprintf(stderr, "%s\n", fileG);
    exit(1);
  }
  
  fp = fopen( fileG, "r");
  
  if( fp == NULL) {  // Make sure input file exits 
    fprintf(stderr, "Error opening loop data file: %s\n", fileG);
    exit(1);  
  }
  
  fgets( line, MAXLINE, fp);
  while( line[0] == '>') {
    fgets( line, MAXLINE, fp);
  }
  
  //Read in Stacking data
  for( i = 0; i < 6; i++) {
    
    nRead = 0;
    token = strtok( line, " ");
    int tmp_array;
    while( token != NULL) {
      tmp_array=0;
      if( sscanf( token, "%d", &tmp_array ) == 1) {
        array[nRead]=tmp_array;
        nRead++;
      }
      token = strtok( NULL, " ");
    }
    
    if( nRead != 6) {
      fprintf(stderr, "Error in stacking data format\n");
      exit(1);
    }
    for( j = 0; j < 6; j++) {
      em->Stack[i*6+j] = G_Stack[i*6+j] = (DBL_TYPE) array[j]/100.0;
    }
    
    fgets( line, MAXLINE, fp);
  }
  
  for( i = 0; i < 3; i++) {
    while( line[0] == '>') {
      fgets( line, MAXLINE, fp);
    }
    
    nRead = 0;
    token = strtok( line, " ");
    while( token != NULL) {
      if( sscanf( token, "%d", &(array[ nRead]) )==1) {
        nRead++;
      }
      token = strtok( NULL, " ");
    }
    
    if( nRead > 30) {
      fprintf(stderr, "Error in Loop energies data\n");
      exit(1);
    }
    for( j = 0; j < 30; j++) {
      if( nRead-1 >= j) {
        em->loop37[30*(2-i)+j] = G_loop37[30*(2-i)+j] = (DBL_TYPE) array[j]/100.0;
      }
      else {
        em->loop37[30*(2-i)+j] = G_loop37[30*(2-i)+j] = G_loop37[30*(2-i)+nRead-1]+
          1.75*kB*temp_k*LOG_FUNC( (j+1)/(1.0*nRead));
      }
    }
    
    fgets( line, MAXLINE, fp);
  }
  
  
  
  while( line[0] == '>') {
    fgets( line, MAXLINE, fp);
  }
  
  nRead = 0;
  token = strtok( line, " ");
  while( token != NULL) {
    if( sscanf( token, "%d", &(array[ nRead]) )==1 ) {
      nRead++;
    }
    token = strtok( NULL, " ");
  }
  
  if( nRead != 5) {
    fprintf(stderr, "Error in asymmetry terms!\n");
    exit(1);
  }
  
  for( j = 0; j < 4; j++) {
    em->asymmetry_penalty[j] = G_asymmetry_penalty[j] = (DBL_TYPE) array[j]/100.0;
  }
  em->max_asymmetry = G_max_asymmetry = (DBL_TYPE) array[4]/100.0;
  
  //Triloops
  fgets( line, MAXLINE, fp);
  while( line[0] == '>') {
    fgets( line, MAXLINE, fp);
  }
  for( i = 0; i < 2048; i++) {
    em->triloop_energy[i] = G_triloop_energy[i] = 0;
  }
  while( line[0] != '>') {
    
    if( sscanf( line, "%s %d", triloop, &(array[0]) ) == 2) {
      indexL = 0;
      for( i = 0 ; i < 5; i++) {
        tmpIndex = 1;
        for( j = 0; j < i; j++) {
          tmpIndex *= 4;
        }
        if( triloop[4-i] == 'C') {
          indexL += tmpIndex;
        }
        else if( triloop[4-i] == 'G') {
          indexL += tmpIndex*2;
        }
        else if( triloop[4-i] == 'U' || triloop[4-i] == 'T') {
          indexL += tmpIndex*3;
        }
        else if( triloop[4-i] != 'A') {
          fprintf(stderr, "Error in triloop indexing %s\n", triloop);
        }
      }
      em->triloop_energy[ indexL] = G_triloop_energy[ indexL] = (DBL_TYPE) array[0]/100.0;
    }
    else {
      fprintf(stderr, "Error in triloop data\n%s\n",line);
    }
    fgets( line, MAXLINE, fp);
  }
  
  //Tetraloops
  while( line[0] == '>') {
    fgets( line, MAXLINE, fp);
  }
  for( i = 0; i < 4096; i++) {
    em->tloop_energy[i] = G_tloop_energy[i] = 0;
  }
  while( line[0] != '>') {
    if( sscanf( line, "%s %d", tetraloop, &(array[0]) ) == 2) {
      indexL = 0;
      for( i = 0 ; i < 6; i++) {
        tmpIndex = 1;
        for( j = 0; j < i; j++) {
          tmpIndex *= 4;
        }
        if( tetraloop[5-i] == 'C') {
          indexL += tmpIndex;
        }
        else if( tetraloop[5-i] == 'G') {
          indexL += tmpIndex*2;
        }
        else if( tetraloop[5-i] == 'U' || tetraloop[5-i] == 'T') {
          indexL += tmpIndex*3;
        }
        else if( tetraloop[5-i] != 'A') {
          fprintf(stderr, "Error in tetraloop indexing %s\n", tetraloop);
        }
      }
      em->tloop_energy[ indexL] = G_tloop_energy[ indexL] = (DBL_TYPE) array[0]/100.0;
    }
    else {
      fprintf(stderr, "Error in tetraloop data\n%s\n",line);
    }
    fgets( line, MAXLINE, fp);
  }
  
  while( line[0] == '>') {
    fgets( line, MAXLINE, fp);
  }
  
  //fprintf(fp, "Mismatch Hairpin: \n");
  for( i = 0; i < 16; i++) {
    
    nRead = 0;
    token = strtok( line, " ");
    while( token != NULL) {
      if( sscanf( token, "%d", &(array[ nRead]) )==1 ) {
        nRead++;
      }
      token = strtok( NULL, " ");
    }
    
    if( nRead != 6) {
      fprintf(stderr, "Error in mismatch hairpin format! %d\n", nRead);
      exit(1);
    }
    
    for( j = 0; j < 6; j++) {
      em->MMEnergiesHP[ 6*i + j] = G_MMEnergiesHP[ 6*i + j] = (DBL_TYPE) array[j]/100.0;
    }
    
    fgets( line, MAXLINE, fp);
  }
  
  while( line[0] == '>') {
    fgets( line, MAXLINE, fp);
  }
  
  //fprintf(fp, "Mismatch Interior: \n");
  for( i = 0; i < 16; i++) {
    
    nRead = 0;
    token = strtok( line, " ");
    while( token != NULL) {
      if( sscanf( token, "%d", &(array[ nRead]) )==1 ) {
        nRead++;
      }
      token = strtok( NULL, " ");
    }
    
    if( nRead != 6) {
      fprintf(stderr, "Error in mismatch Interior format!\n");
      exit(1);
    }
    
    for( j = 0; j < 6; j++) {
      em->MMEnergiesIL[ 6*i + j] = G_MMEnergiesIL[ 6*i + j] = (DBL_TYPE) array[j]/100.0;
    }
    
    fgets( line, MAXLINE, fp);
  }
  
  while( line[0] == '>') {
    fgets( line, MAXLINE, fp);
  }
  
  //Read in Dangles
  for( i = 0; i < 6; i++) {
    nRead = 0;
    token = strtok( line, " ");
    while( token != NULL) {
      if( sscanf( token, "%d", &(array[ nRead]) )==1 ) {
        nRead++;
      }
      token = strtok( NULL, " ");
    }
    
    if( nRead != 4) {
      fprintf(stderr, "1. Error in dangle data format!\n");
      exit(1);
    }
    for( j = 0; j < 4; j++) {
      em->dangle_energy[i*4+j]  = G_dangle_energy[i*4+j] = (DBL_TYPE) array[j]/100.0;
      
      if( DANGLETYPE == 0)  em->dangle_energy[i*4+j] = G_dangle_energy[i*4+j] = 0.0;

    }
    
    fgets( line, MAXLINE, fp);
  }
  
  while( line[0] == '>') {
    fgets( line, MAXLINE, fp);
  }
  
  //Read in Dangles
  for( i = 0; i < 6; i++) {
    nRead = 0;
    token = strtok( line, " ");
    while( token != NULL) {
      if( sscanf( token, "%d", &(array[ nRead]) )==1 ) {
        nRead++;
      }
      token = strtok( NULL, " ");
    }
    
    if( nRead != 4) {
      fprintf(stderr, "2. Error in dangle data format!\n");
      exit(1);
    }
    for( j = 0; j < 4; j++) {
      em->dangle_energy[24+ i*4+j] = G_dangle_energy[24+ i*4+j] = (DBL_TYPE) array[j]/100.0;
      if( DANGLETYPE == 0) em->dangle_energy[24+ i*4+j] = G_dangle_energy[24+ i*4+j] = 0;
    }
    
    fgets( line, MAXLINE, fp);
  }
  
  while( line[0] == '>') {
    fgets( line, MAXLINE, fp);
  }


  //Multiloop parameters
  nRead = 0;
  token = strtok( line, " ");
  while( token != NULL) {
    if( sscanf( token, "%d", &(array[ nRead]) ) ==1) {
      
      nRead++;
    }
    token = strtok( NULL, " ");
  }
  
  
  if( nRead != 3) {
    fprintf(stderr, "3. Error in dangle data format!\n");
    exit(1);
  }
  
  em->alpha_1 = G_ALPHA_1 = (DBL_TYPE) array[0]/100.0;
  em->alpha_2 = G_ALPHA_2 = (DBL_TYPE) array[1]/100.0;
  em->alpha_3 = G_ALPHA_3 = (DBL_TYPE) array[2]/100.0;
  
  fgets( line, MAXLINE, fp);
  while( line[0] == '>') {
    fgets( line, MAXLINE, fp);
  }
  
  //AT PENALTY
  if( sscanf(line, "%d", &(array[0]) ) == 1) {
    em->at_penalty = G_AT_PENALTY = (DBL_TYPE) array[0]/100.0;
  }
  else {
    fprintf(stderr, "Error in AT PENALTY data\n");
    exit(1);
  }
  
  fgets( line, MAXLINE, fp);
  while( line[0] == '>') {
    fgets( line, MAXLINE, fp);
  }
  
  //1x1 interior loop
  for( i = 0; i < 36; i++) {
    fgets( line, MAXLINE, fp); //read in label
    for( j = 0; j < 4; j++) {
      
      nRead = 0;
      token = strtok( line, " ");
      while( token != NULL) {
        if( sscanf( token, "%d", &(array[ nRead]) )==1 ) {
          
          nRead++;
        }
        token = strtok( NULL, " ");
      }
      
      if( nRead != 4) {
        fprintf(stderr, "Error in 1x1 format!\n");
        exit(1);
      }
      
      for( k = 0; k < 4; k++) {
        em->IL_SInt2[ i*16 + j*4 + k] = G_IL_SInt2[ i*16 + j*4 + k] = (DBL_TYPE) array[ k]/100.0;
      }
      
      fgets( line, MAXLINE, fp);
    }
  }
  
  
  while( line[0] == '>') {
    fgets( line, MAXLINE, fp);
  }
  
  //2x2 interior loop
  for( i = 0; i < 36*16; i++) {
    fgets( line, MAXLINE, fp); //read in label
    for( j = 0; j < 4; j++) {
      
      nRead = 0;
      token = strtok( line, " ");
      while( token != NULL) {
        if( sscanf( token, "%d", &(array[ nRead]) )==1 ) {
          
          nRead++;
        }
        token = strtok( NULL, " ");
      }
      
      if( nRead != 4) {
        fprintf(stderr, "Error in 1x1 format!\n");
        exit(1);
      }
      
      for( k = 0; k < 4; k++) {
        em->IL_SInt4[ 1536*(i/96)+256*((i%96)/16) +
                 64*((i%16)/4) + 4*(i%4) + k*16 + j] = 
          G_IL_SInt4[ 1536*(i/96)+256*((i%96)/16) +
                     64*((i%16)/4) + 4*(i%4) + k*16 + j] = 
          (DBL_TYPE) array[ k]/100.0;
      }
      
      fgets( line, MAXLINE, fp);
    }
  }
  
  while( line[0] == '>') {
    fgets( line, MAXLINE, fp);
  }
  
  //1x2 interior loop
  for( i = 0; i < 144; i++) {
    fgets( line, MAXLINE, fp); //read in label
    for( j = 0; j < 4; j++) {
      
      nRead = 0;
      token = strtok( line, " ");
      while( token != NULL) {
        if( sscanf( token, "%d", &(array[ nRead]) )==1 ) {
          
          nRead++;
        }
        token = strtok( NULL, " ");
      }
      
      if( nRead != 4) {
        fprintf(stderr, "Error in 1x1 format!\n");
        exit(1);
      }
      
      for( k = 0; k < 4; k++) {
        em->IL_AsInt1x2[ 384*(i/24) + 4*((i%24)/4) + 24*(i%4) +
                    96*j + k] = G_IL_AsInt1x2[ 384*(i/24) + 4*((i%24)/4) + 24*(i%4) +
                                              96*j + k] = (DBL_TYPE) array[ k]/100.0;
      }
      
      fgets( line, MAXLINE, fp);
    }
  }
  
  while( line[0] == '>') {
    fgets( line, MAXLINE, fp);
  }
  
  //polyC hairpin parameters
  nRead = 0;

  token = strtok( line, " ");
  while( token != NULL) {
    if( sscanf( token, "%d", &(array[ nRead]) )==1 ) {
      
      nRead++;
    }
    token = strtok( NULL, " ");
  }
  
  if( nRead != 3) {
    fprintf(stderr, "4. Error in polyC hairpin parameters!\n");
    exit(1);
  }
  
  em->polyc3 = G_POLYC3 = (DBL_TYPE) array[0]/100.0;
  em->polycslope = G_POLYCSLOPE = (DBL_TYPE) array[1]/100.0;
  em->polycint = G_POLYCINT = (DBL_TYPE) array[2]/100.0;
  
  fgets( line, MAXLINE, fp);
  while( line[0] == '>') {
    fgets( line, MAXLINE, fp);
  }
  
  
  //Pseudoknot parameters
  nRead = 0;
  token = strtok( line, " ");
  while( token != NULL) {
    if( sscanf( token, "%d", &(array[ nRead]) )==1 ) {
      
      nRead++;
    }
    token = strtok( NULL, " ");
  }
  
  if( nRead != 5) {
    fprintf(stderr, "5. Error in dangle data format!\n");
    exit(1);
  }
  
  em->beta_1 = G_BETA_1 = (DBL_TYPE) array[0]/100.0;
  em->beta_2 = G_BETA_2 = (DBL_TYPE) array[1]/100.0;
  em->beta_3 = G_BETA_3 = (DBL_TYPE) array[2]/100.0;
  em->beta_1m = G_BETA_1M = (DBL_TYPE) array[3]/100.0;
  em->beta_1p = G_BETA_1P = (DBL_TYPE) array[4]/100.0;
  
  fgets( line, MAXLINE, fp);
  while( line[0] == '>') {
    fgets( line, MAXLINE, fp);
  }
  
  //BIMOLECULAR TERM
  nRead = 0;
  token = strtok( line, " ");
  while( token != NULL) {
    if( sscanf( token, "%d", &(array[ nRead]) )==1 ) {
      
      nRead++;
    }
    token = strtok( NULL, " ");
  }
  
  if( nRead != 1) {
    fprintf(stderr, "Error in BIMOLECULAR format!\n");
    exit(1);
  }
  
  G_BIMOLECULAR = (DBL_TYPE) array[0]/100.0;
  em->bimolecular = G_BIMOLECULAR - kB*temp_k*LOG_FUNC( water_conc);
  
  fclose( fp);
  
  /* ****************************** */
  //Load Enthalpies and calculate modified G
  
  //If Temperature == 37 C, add the salt correction andskip he dH file.
  //  Else, make sure it is present
  if( temp_k > 37.0 + ZERO_C_IN_KELVIN - 0.001 && temp_k < 37.0 + ZERO_C_IN_KELVIN + 0.001) {

    // Make the salt corrections
    // Stacked bases
    for (i = 0; i < 36; i++) {
      em->Stack[i] += em->salt_correction;
    }
    
    // Loop correction.  Covers all hairpins, and bulges, but overcounts for
    // 1-base bulges.  This is corrected in the function InteriorEnergyFull.
    // This also covers large (non-tabulated) interior loops.
    for (i = 0; i < 90; i++) {
      em->loop37[i] += em->salt_correction;
    }
    
    // Corrections for tabulated interior loops
    for (i = 0; i < 16*36; i++) {
      em->IL_SInt2[i] += em->salt_correction;
    }
    for (i = 0; i < 256*36; i++) {
      em->IL_SInt4[i] += em->salt_correction;
    }
    for (i = 0; i < 64*36; i++) {
      em->IL_AsInt1x2[i] += em->salt_correction;
    }
    
    // Multiloop
    em->alpha_1 += em->salt_correction;
    
    return;
  }
  
  //check first for parameter files using current directory as home
  strcpy( fileH, fileNameRoot);
  strcat( fileH, ".dH");
  
  if( !fileExists( fileH) ) {
    //if files not found, use environment variable NUPACKHOME as root
    nupackhome = getenv("NUPACKHOME");
    if( nupackhome != NULL) {
      
      strcpy( fileH, nupackhome);
      strcat( fileH, "/parameters/");
      strcat( fileH, fileNameRoot);
      strcat( fileH, ".dH");
    }
    else {
      fprintf(stderr, "Unable to find %s.dH locally, and NUPACKHOME environment variable is not set.\n",
             fileNameRoot);
      fprintf(stderr, "Consequently, your temperature must be set to 37.0 C, not %.1f.  Job Aborted.\n", 
             (float) (temp_k - ZERO_C_IN_KELVIN));
      exit(1);
    }
  }
  
  if( ! fileExists( fileH)) {
    fprintf(stderr, "Unable to find file %s.dH locally or in NUPACKHOME = %s\n", fileNameRoot,
           nupackhome);
    fprintf(stderr, "Consequently, your temperature must be set to 37.0 C, not %.1f.  Job Aborted.\n",
           (float) (temp_k - ZERO_C_IN_KELVIN) );
    exit(1);
  }
  
  
  fp = fopen( fileH, "r");
  if( fp == NULL) {  // Make sure input file exits 
    fprintf(stderr, "Error opening loop data file: %s\n", fileH);
    exit(1);  
  }
  
  fgets( line, MAXLINE, fp);
  while( line[0] == '>') {
    fgets( line, MAXLINE, fp);
  }
  
  //Read in Stacking data
  for( i = 0; i < 6; i++) {
    
    nRead = 0;
    token = strtok( line, " ");
    while( token != NULL) {
      if( sscanf( token, "%d", &(array[ nRead]) ) == 1) {
        nRead++;
      }
      token = strtok( NULL, " ");
    }
    
    if( nRead != 6) {
      fprintf(stderr, "Error in stacking data format\n");
      exit(1);
    }
    for( j = 0; j < 6; j++) {
      H_Stack[i*6+j] = (DBL_TYPE) array[j]/100.0;
      em->Stack[i*6+j] = (G_Stack[i*6+j] - H_Stack[i*6+j])*temp_k/310.15
        + H_Stack[i*6+j];
    }
    
    fgets( line, MAXLINE, fp);
  }
  
  for( i = 0; i < 3; i++) {
    while( line[0] == '>') {
      fgets( line, MAXLINE, fp);
    }
    
    nRead = 0;
    token = strtok( line, " ");
    while( token != NULL) {
      if( sscanf( token, "%d", &(array[ nRead]) )==1) {
        nRead++;
      }
      token = strtok( NULL, " ");
    }
    
    if( nRead > 30) {
      fprintf(stderr, "Error in Loop energies data\n");
      exit(1);
    }
    for( j = 0; j < 30; j++) {
      if( nRead-1 >= j) {
        H_loop37[30*(2-i)+j] = (DBL_TYPE) array[j]/100.0;
      }
      else {
        H_loop37[30*(2-i)+j] = H_loop37[30*(2-i)+nRead-1]+
          1.75*kB*temp_k*LOG_FUNC( (j+1)/(1.0*nRead));
      }
      
      em->loop37[30*(2-i)+j] = (G_loop37[30*(2-i)+j] - H_loop37[30*(2-i)+j])*
        temp_k/310.15 + H_loop37[30*(2-i)+j];
    }
    
    fgets( line, MAXLINE, fp);
  }
  
  while( line[0] == '>') {
    fgets( line, MAXLINE, fp);
  }
  
  nRead = 0;
  token = strtok( line, " ");
  while( token != NULL) {
    if( sscanf( token, "%d", &(array[ nRead]) )==1 ) {
      nRead++;
    }
    token = strtok( NULL, " ");
  }
  
  if( nRead != 5) {
    fprintf(stderr, "Error in asymmetry terms!\n");
    exit(1);
  }
  
  for( j = 0; j < 4; j++) {
    H_asymmetry_penalty[j] = (DBL_TYPE) array[j]/100.0;
    em->asymmetry_penalty[j] = (G_asymmetry_penalty[j] - 
                            H_asymmetry_penalty[j])*
      temp_k/310.15 + H_asymmetry_penalty[j];
  }
  H_max_asymmetry = (DBL_TYPE) array[4]/100.0;
  em->max_asymmetry = (G_max_asymmetry - 
                   H_max_asymmetry)* temp_k/310.15 + H_max_asymmetry;
  
  
  //Triloops
  fgets( line, MAXLINE, fp);
  while( line[0] == '>') {
    fgets( line, MAXLINE, fp);
  }
  for( i = 0; i < 2048; i++) {
    H_triloop_energy[i] = 0;
    em->triloop_energy[i] = (G_triloop_energy[i] - 
                         H_triloop_energy[i])* temp_k/310.15 + 
      H_triloop_energy[i];
  }
  while( line[0] != '>') {
    
    if( sscanf( line, "%s %d", triloop, &(array[0]) ) == 2) {
      indexL = 0;
      for( i = 0 ; i < 5; i++) {
        tmpIndex = 1;
        for( j = 0; j < i; j++) {
          tmpIndex *= 4;
        }
        if( triloop[4-i] == 'C') {
          indexL += tmpIndex;
        }
        else if( triloop[4-i] == 'G') {
          indexL += tmpIndex*2;
        }
        else if( triloop[4-i] == 'U' || triloop[4-i] == 'T') {
          indexL += tmpIndex*3;
        }
        else if( triloop[4-i] != 'A') {
          fprintf(stderr, "Error in triloop indexing %s\n", triloop);
        }
      }
      H_triloop_energy[ indexL] = (DBL_TYPE) array[0]/100.0;
      em->triloop_energy[indexL] = (G_triloop_energy[indexL] - 
                                H_triloop_energy[indexL])* temp_k/310.15 + 
        H_triloop_energy[indexL];
    }
    else {
      fprintf(stderr, "Error in triloop data\n%s\n",line);
    }
    fgets( line, MAXLINE, fp);
  }
  
  //Tetraloops
  while( line[0] == '>') {
    fgets( line, MAXLINE, fp);
  }
  for( i = 0; i < 4096; i++) {
    H_tloop_energy[i] = 0;
    em->tloop_energy[i] = (G_tloop_energy[i] - 
                       H_tloop_energy[i])* temp_k/310.15 + 
      H_tloop_energy[i];
    //printf("%f ", temp_k);
  }
  while( line[0] != '>') {
    
    if( sscanf( line, "%s %d", tetraloop, &(array[0]) ) == 2) {
      indexL = 0;
      for( i = 0 ; i < 6; i++) {
        tmpIndex = 1;
        for( j = 0; j < i; j++) {
          tmpIndex *= 4;
        }
        if( tetraloop[5-i] == 'C') {
          indexL += tmpIndex;
        }
        else if( tetraloop[5-i] == 'G') {
          indexL += tmpIndex*2;
        }
        else if( tetraloop[5-i] == 'U' || tetraloop[5-i] == 'T') {
          indexL += tmpIndex*3;
        }
        else if( tetraloop[5-i] != 'A') {
          fprintf(stderr, "Error in tetraloop indexing %s\n", tetraloop);
        }
      }
      
      H_tloop_energy[ indexL] = (DBL_TYPE) array[0]/100.0;
      em->tloop_energy[indexL] = (G_tloop_energy[indexL] - 
                              H_tloop_energy[indexL])* temp_k/310.15 + 
        H_tloop_energy[indexL];
    }
    else {
      fprintf(stderr, "Error in tetraloop data\n%s\n",line);
    }
    fgets( line, MAXLINE, fp);
  }
  
  while( line[0] == '>') {
    fgets( line, MAXLINE, fp);
  }
  
  //fprintf(fp, "Mismatch Hairpin: \n");
  for( i = 0; i < 16; i++) {
    
    nRead = 0;
    token = strtok( line, " ");
    while( token != NULL) {
      if( sscanf( token, "%d", &(array[ nRead]) )==1 ) {
        nRead++;
      }
      token = strtok( NULL, " ");
    }
    
    if( nRead != 6) {
      fprintf(stderr, "Error in mismatch hairpin format! %d\n", nRead);
      exit(1);
    }
    
    for( j = 0; j < 6; j++) {
      H_MMEnergiesHP[ 6*i + j] = (DBL_TYPE) array[j]/100.0;
      em->MMEnergiesHP[6*i+j] = (G_MMEnergiesHP[6*i+j] - 
                             H_MMEnergiesHP[6*i+j])* temp_k/310.15 + 
        H_MMEnergiesHP[6*i+j];
    }
    
    fgets( line, MAXLINE, fp);
  }
  
  while( line[0] == '>') {
    fgets( line, MAXLINE, fp);
  }
  
  //fprintf(fp, "Mismatch Interior: \n");
  for( i = 0; i < 16; i++) {
    
    nRead = 0;
    token = strtok( line, " ");
    while( token != NULL) {
      if( sscanf( token, "%d", &(array[ nRead]) )==1 ) {
        nRead++;
      }
      token = strtok( NULL, " ");
    }
    
    if( nRead != 6) {
      fprintf(stderr, "Error in mismatch Interior format!\n");
      exit(1);
    }
    
    for( j = 0; j < 6; j++) {
      H_MMEnergiesIL[ 6*i + j] = (DBL_TYPE) array[j]/100.0;
      em->MMEnergiesIL[6*i+j] = (G_MMEnergiesIL[6*i+j] - 
                             H_MMEnergiesIL[6*i+j])* temp_k/310.15 + 
        H_MMEnergiesIL[6*i+j];
    }
    
    fgets( line, MAXLINE, fp);
  }
  
  while( line[0] == '>') {
    fgets( line, MAXLINE, fp);
  }
  
  //Read in Dangles
  for( i = 0; i < 6; i++) {
    nRead = 0;
    token = strtok( line, " ");
    while( token != NULL) {
      if( sscanf( token, "%d", &(array[ nRead]) )==1 ) {
        nRead++;
      }
      token = strtok( NULL, " ");
    }
    
    if( nRead != 4) {
      fprintf(stderr, "6. Error in dangle data format!\n");
      exit(1);
    }
    for( j = 0; j < 4; j++) {
      H_dangle_energy[i*4+j] = (DBL_TYPE) array[j]/100.0;
      if( DANGLETYPE == 0) H_dangle_energy[i*4+j] = 0;
      
      em->dangle_energy[i*4+j] = (G_dangle_energy[i*4+j] - 
                              H_dangle_energy[i*4+j])* temp_k/310.15 + 
        H_dangle_energy[i*4+j];
    }
    
    fgets( line, MAXLINE, fp);
  }
  
  while( line[0] == '>') {
    fgets( line, MAXLINE, fp);
  }
  
  //Read in Dangles
  for( i = 0; i < 6; i++) {
    nRead = 0;
    token = strtok( line, " ");
    while( token != NULL) {
      if( sscanf( token, "%d", &(array[ nRead]) )==1 ) {
        nRead++;
      }
      token = strtok( NULL, " ");
    }
    
    if( nRead != 4) {
      fprintf(stderr, "7. Error in dangle data format!\n");
      exit(1);
    }
    for( j = 0; j < 4; j++) {
      H_dangle_energy[24+ i*4+j] = (DBL_TYPE) array[j]/100.0;
      if( DANGLETYPE == 0) 
        H_dangle_energy[24+ i*4+j] = 0;
      
      em->dangle_energy[24+i*4+j] = (G_dangle_energy[24+i*4+j] - 
                                 H_dangle_energy[24+i*4+j])* temp_k/310.15 + 
        H_dangle_energy[24+i*4+j];
      
    }
    
    fgets( line, MAXLINE, fp);
  }
  
  while( line[0] == '>') {
    fgets( line, MAXLINE, fp);
  }
  //Multiloop parameters
  nRead = 0;
  token = strtok( line, " ");
  while( token != NULL) {
    if( sscanf( token, "%d", &(array[ nRead]) ) ==1) {
      
      nRead++;
    }
    token = strtok( NULL, " ");
  }
  
  
  if( nRead != 3) {
    fprintf(stderr, "8. Error in dangle data format!\n");
    exit(1);
  }
  
  H_ALPHA_1 = (DBL_TYPE) array[0]/100.0;
  H_ALPHA_2 = (DBL_TYPE) array[1]/100.0;
  H_ALPHA_3 = (DBL_TYPE) array[2]/100.0;
  
  em->alpha_1 = (G_ALPHA_1 - H_ALPHA_1)* temp_k/310.15 + 
    H_ALPHA_1;
  em->alpha_2 = (G_ALPHA_2 - H_ALPHA_2)* temp_k/310.15 + 
    H_ALPHA_2;
  em->alpha_3 = (G_ALPHA_3 - H_ALPHA_3)* temp_k/310.15 + 
    H_ALPHA_3;
  
  
  fgets( line, MAXLINE, fp);
  while( line[0] == '>') {
    fgets( line, MAXLINE, fp);
  }
  
  //AT PENALTY
  if( sscanf(line, "%d", &(array[0]) ) == 1) {
    H_AT_PENALTY = (DBL_TYPE) array[0]/100.0;
    em->at_penalty = (G_AT_PENALTY - H_AT_PENALTY)* temp_k/310.15 + 
      H_AT_PENALTY;
    
  }
  else {
    fprintf(stderr, "Error in AT PENALTY data\n");
    exit(1);
  }
  
  fgets( line, MAXLINE, fp);
  while( line[0] == '>') {
    fgets( line, MAXLINE, fp);
  }
  
  //1x1 interior loop
  for( i = 0; i < 36; i++) {
    fgets( line, MAXLINE, fp); //read in label
    for( j = 0; j < 4; j++) {
      
      nRead = 0;
      token = strtok( line, " ");
      while( token != NULL) {
        if( sscanf( token, "%d", &(array[ nRead]) )==1 ) {
          
          nRead++;
        }
        token = strtok( NULL, " ");
      }
      
      if( nRead != 4) {
        fprintf(stderr, "Error in 1x1 format!\n");
        exit(1);
      }
      
      for( k = 0; k < 4; k++) {
        H_IL_SInt2[ i*16 + j*4 + k] = (DBL_TYPE) array[ k]/100.0;
        em->IL_SInt2[i*16+j*4+k] = (G_IL_SInt2[i*16+j*4+k] - 
                                H_IL_SInt2[i*16+j*4+k])* temp_k/310.15 + 
          H_IL_SInt2[i*16+j*4+k];
        
      }
      
      fgets( line, MAXLINE, fp);
    }
  }
  
  
  while( line[0] == '>') {
    fgets( line, MAXLINE, fp);
  }
  
  //2x2 interior loop
  for( i = 0; i < 36*16; i++) {
    fgets( line, MAXLINE, fp); //read in label
    for( j = 0; j < 4; j++) {
      
      nRead = 0;
      token = strtok( line, " ");
      while( token != NULL) {
        if( sscanf( token, "%d", &(array[ nRead]) )==1 ) {
          
          nRead++;
        }
        token = strtok( NULL, " ");
      }
      
      if( nRead != 4) {
        fprintf(stderr, "Error in 1x1 format!\n");
        exit(1);
      }
      
      for( k = 0; k < 4; k++) {
        index4 = 1536*(i/96)+256*((i%96)/16) +
          64*((i%16)/4) + 4*(i%4) + k*16 + j;
        H_IL_SInt4[index4] = 
          (DBL_TYPE) array[ k]/100.0;
        em->IL_SInt4[index4] = 
          (G_IL_SInt4[index4] - 
           H_IL_SInt4[index4])* temp_k/310.15 + 
          H_IL_SInt4[index4];
        
      }
      
      fgets( line, MAXLINE, fp);
    }
  }
  
  while( line[0] == '>') {
    fgets( line, MAXLINE, fp);
  }
  
  //1x2 interior loop
  for( i = 0; i < 144; i++) {
    fgets( line, MAXLINE, fp); //read in label
    for( j = 0; j < 4; j++) {
      
      nRead = 0;
      token = strtok( line, " ");
      while( token != NULL) {
        if( sscanf( token, "%d", &(array[ nRead]) )==1 ) {
          
          nRead++;
        }
        token = strtok( NULL, " ");
      }
      
      if( nRead != 4) {
        fprintf(stderr, "Error in 1x1 format!\n");
        exit(1);
      }
      
      for( k = 0; k < 4; k++) {
        index4 = 384*(i/24) + 4*((i%24)/4) + 24*(i%4) +
          96*j + k;
        H_IL_AsInt1x2[index4] = (DBL_TYPE) array[ k]/100.0;
        em->IL_AsInt1x2[index4] = 
          (G_IL_AsInt1x2[index4] - 
           H_IL_AsInt1x2[index4])* temp_k/310.15 + 
          H_IL_AsInt1x2[index4];
      }
      
      fgets( line, MAXLINE, fp);
    }
  }
  
  while( line[0] == '>') {
    fgets( line, MAXLINE, fp);
  }
  
  //polyC hairpin parameters
  nRead = 0;
  token = strtok( line, " ");
  while( token != NULL) {
    if( sscanf( token, "%d", &(array[ nRead]) )==1 ) {
      
      nRead++;
    }
    token = strtok( NULL, " ");
  }
  
  if( nRead != 3) {
    fprintf(stderr, "9. Error in dangle data format!\n");
    exit(1);
  }
  
  H_POLYC3 = (DBL_TYPE) array[0]/100.0;
  H_POLYCSLOPE = (DBL_TYPE) array[1]/100.0;
  H_POLYCINT = (DBL_TYPE) array[2]/100.0;
  
  em->polyc3 = (G_POLYC3 - H_POLYC3)* temp_k/310.15 + 
    H_POLYC3;
  em->polycslope = (G_POLYCSLOPE - H_POLYCSLOPE)* temp_k/310.15 + 
    H_POLYCSLOPE;
  em->polycint = (G_POLYCINT - H_POLYCINT)* temp_k/310.15 + H_POLYCINT;
  
  
  fgets( line, MAXLINE, fp);
  while( line[0] == '>') {
    fgets( line, MAXLINE, fp);
  }
  
  //Pseudoknot parameters
  nRead = 0;
  token = strtok( line, " ");
  while( token != NULL) {
    if( sscanf( token, "%d", &(array[ nRead]) )==1 ) {
      
      nRead++;
    }
    token = strtok( NULL, " ");
  }
  
  if( nRead != 5) {
    fprintf(stderr, "Error in pseudoknot data format!\n");
    exit(1);
  }
  
  H_BETA_1 = (DBL_TYPE) array[0]/100.0;
  H_BETA_2 = (DBL_TYPE) array[1]/100.0;
  H_BETA_3 = (DBL_TYPE) array[2]/100.0;
  H_BETA_1M = (DBL_TYPE) array[3]/100.0;
  H_BETA_1P = (DBL_TYPE) array[4]/100.0;
  
  em->beta_1 = (G_BETA_1 - H_BETA_1)* temp_k/310.15 + 
    H_BETA_1;
  em->beta_2 = (G_BETA_2 - H_BETA_2)* temp_k/310.15 + 
    H_BETA_2;
  em->beta_3 = (G_BETA_3 - H_BETA_3)* temp_k/310.15 + 
    H_BETA_3;
  em->beta_1m = (G_BETA_1M - H_BETA_1M)* temp_k/310.15 + 
    H_BETA_1M;
  em->beta_1p = (G_BETA_1P - H_BETA_1P)* temp_k/310.15 + 
    H_BETA_1P;
  
  fgets( line, MAXLINE, fp);
  while( line[0] == '>') {
    fgets( line, MAXLINE, fp);
  }
  
  //BIMOLECULAR TERM
  nRead = 0;
  token = strtok( line, " ");
  while( token != NULL) {
    if( sscanf( token, "%d", &(array[ nRead]) )==1 ) {
      
      nRead++;
    }
    token = strtok( NULL, " ");
  }
  
  if( nRead != 1) {
    fprintf(stderr, "Error in bimolecular data format!\n");
    exit(1);
  }
  
  H_BIMOLECULAR = (DBL_TYPE) array[0]/100.0;
  
  em->bimolecular = (G_BIMOLECULAR - H_BIMOLECULAR)* temp_k/310.15 + 
    H_BIMOLECULAR - kB*temp_k*LOG_FUNC( water_conc);
  
  fclose( fp);


  // Make the salt corrections
  // Stacked bases
  for (i = 0; i < 36; i++) {
    em->Stack[i] += em->salt_correction;
  }

  // Loop correction.  Covers all hairpins, and bulges, but overcounts for
  // 1-base bulges.  This is corrected in the function InteriorEnergyFull.
  // This also covers large (non-tabulated) interior loops.
  for (i = 0; i < 90; i++) {
    em->loop37[i] += em->salt_correction;
  }

  // Corrections for tabulated interior loops
  for (i = 0; i < 16*36; i++) {
    em->IL_SInt2[i] += em->salt_correction;
  }
  for (i = 0; i < 256*36; i++) {
    em->IL_SInt4[i] += em->salt_correction;
  }
  for (i = 0; i < 64*36; i++) {
    em->IL_AsInt1x2[i] += em->salt_correction;
  }

  // Multiloop
  em->alpha_1 += em->salt_correction;

}
#undef fgets

/* ************** */

void setParametersToZero(energy_model_t *em) {
  
  int i;
	
  for( i = 0; i < 90; i++) {
    em->loop37[i] = 0;
  }
  for ( i = 0; i < 4096; i++) {
    em->tloop_energy[i] = 0;
  }
  for( i = 0; i < 2048; i++) {
    em->triloop_energy[i] = 0;
  }
  for( i = 0; i < 16*6; i++) {
    em->MMEnergiesHP[i] = 0;
  }
  for( i = 0; i < 256; i++) {
    em->MMEnergiesIL[i] = 0;
  }
  for( i = 0; i < 16*36; i++) {
    em->IL_SInt2[i] = 0;
  }
  for( i = 0; i < 256*36; i++) {
    em->IL_SInt4[ i] = 0;
  }
  for( i = 0; i < 64*36; i++) {
    em->IL_AsInt1x2[i] = 0;
  }
  for( i = 0; i < 48; i++) {
    em->dangle_energy[i] = 0;
  }
  for( i = 0; i < 4; i++) {
    em->asymmetry_penalty[i] = 0;
  }
  em->max_asymmetry = 0;
	
  for( i = 0; i < 36; i++) {
    em->Stack[i] = 0;
  }

  em->alpha_1 = em->alpha_2 = em->alpha_3 = em->beta_1 = em->beta_2 =
    em->beta_3 = em->beta_1m = em->beta_1p = em->polyc3 =
    em->polycint = em->polycslope = em->at_penalty = em-> bimolecular = 0;
}

/* *************** */
DEV
void InitEtaN( int **etaN, const int *nicks, int seqlength) {
  
  int i,j,k, nick;
  int indexE;
  
  for( i = 0; i <= seqlength-1; i++) {
    for( j = i-1; j <= seqlength-1; j++) {
      indexE = pf_index( i, j, seqlength);
      /*
      etaN[ indexE] = (int *) malloc( 2*sizeof( int));
      if(!etaN[indexE]) {
        printf("etaN allocation failed\n");
        assert(0);
      }
      */
      etaN[ indexE][0] = 0;
      etaN[ indexE][1] = -1;
      
    }
  }
  
  k = 0;
  nick = nicks[k];
  while( nick != -1) {
    for( i = 0; i <= nick; i++) {
      for( j = nick; j <= seqlength-1; j++) { 
        indexE =  pf_index(i,j,seqlength);
        etaN[ indexE][0]++;
        if( etaN[ indexE][1] == -1) { 
          //assume nicks are assigned in increasing order 
          etaN[indexE][1] = k;
        }
      }
    }
    nick = nicks[++k];
  }
}

/* *************************** */
int EtaNIndex_old( float i, float j, int seqlength) { 
  return pf_index( (int) i, (int) j, seqlength);
}

/* ******************* */


