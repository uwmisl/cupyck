#include "hip/hip_runtime.h"
/*
  ene.c is part of the NUPACK software suite
  Copyright (c) 2007 Caltech. All rights reserved.
  Coded by: Robert Dirks 3/2006, Justin Bois 1/2007

  This file contains energy functions used for determining energies
*/


#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <assert.h>
#include <string.h>
#include <time.h>
#include <float.h>

#include "pfuncUtilsHeader.h"
#include "DNAExternals.h"

DEV DBL_TYPE ExplDangleRaw( int i, int j, int seq[], int seqlength);

/* ************************************** */
DEV
DBL_TYPE HelixEnergy( int i, int j, int h, int m, energy_model_t *em) {
  // Calculate the energy of the helical region closed by pair
  // i-j and h-m.  Data from Zuker's mfold file stack.dgd

  int shift_ij; // Type of base pair
  int shift_hm; // Type of base pair

  shift_ij = GetMismatchShift( i, j);
  shift_hm = GetMismatchShift( h, m);

  if( shift_ij < 4 && shift_hm < 4) {
    return em->Stack[ ( i - 1)*6 + (h - 1) ];
  }

  if( shift_ij < 4 && shift_hm >= 4) {
    return em->Stack[ (i - 1)*6 + (h + 1) ];
  }

  if( shift_ij >= 4 && shift_hm < 4) {
    return em->Stack[ (i + 1)*6 + (h - 1) ];
  }

  if( shift_ij >= 4 && shift_hm >= 4) {
    return em->Stack[ (i + 1)*6 + (h + 1) ];
  }
  else {
    printf("Error in HelixEnergy!");
    assert(0);
    return NAD_INFINITY; // This never is returned
  }
}

// *******************************************************************
DEV
DBL_TYPE InteriorMM( char a, char b, char x, char y, energy_model_t *em) {
/*
  Interior Mismatch calculation

  This calculates the Mismatch interaction energies between positions
  1 -> 5' a x 3'
  2 -> 3' b y 5'
  Interactions energies taken from file tstacki2.dgd.
*/

  int cp_shift;
  DBL_TYPE energy = 0.0;

  cp_shift = GetMismatchShift( a, b );
  energy = em->MMEnergiesIL[ (4*(( x) - 1) + (( y) - 1) )*6 + cp_shift];

  return energy;
}

/* ********************************************** */

DEV
DBL_TYPE HairpinEnergy( int i, int j, int seq[], energy_model_t *em) {

  // This gives the energy of the hairpion closed by bases i and j
  DBL_TYPE energy;  //energy of hairpin

  int triloopnumber; //Index for specific triloop
  int tloopnumber; //index for tloops

  int size; //Doesn't include closing pair i-j

  int cp_shift; //Classification of base-pair for energy mismatch

  int polyC = TRUE;  //Is the hairpin a poly-C?
  int k;
  for( k = i+1; k < j; k++) {
    if( seq[k] != BASE_C) {
      polyC = FALSE;
      break;
    }
  }

  size = j - i - 1;

  if( size < 3) {
    return NAD_INFINITY;
  }

  if( CanPair( seq[i], seq[j]) == FALSE ) {
    return NAD_INFINITY;
  }

  if( size <= 30) {
    energy = em->loop37[ 60 + size - 1];
  }
  else {
    energy = em->loop37[ 60 + 30 - 1];
    energy += sizeLog (size, em); //1.75*kB*TEMP_K*LOG_FUNC( size/30.0);

    if( em->dnarnacount == COUNT) {
      energy = 0;
    }

  }

  if( size == 3) {
    //Get Triloop energy

    if( seq[i] != BASE_C && seq[j] != BASE_C) {
      energy += em->at_penalty;
    }

    triloopnumber = 256*(( seq[i]) - 1) +
      64*(( seq[i + 1]) - 1) +
      16*(( seq[i + 2]) - 1) +
      4*( ( seq[j - 1]) - 1) +
      1*( ( seq[j]) - 1);

    // 0 mismatch energy for triloops
    energy += em->triloop_energy[ triloopnumber];

    //Poly-C loop
    if( polyC == TRUE) {
      energy += em->polyc3;
    }
  }
  else if (size == 4) {

    tloopnumber = 1024*(( seq[i]) - 1) +
      256*(( seq[i + 1]) - 1) +
      64*( ( seq[i + 2]) - 1) +
      16*( ( seq[j - 2]) - 1) +
      4*(  ( seq[j - 1]) - 1) +
      1*(  ( seq[j])- 1);
    energy +=  em->tloop_energy[ tloopnumber];

    //Next do mismatches.
    cp_shift = GetMismatchShift( seq[i], seq[j]);

    energy += em->MMEnergiesHP[(4*(( seq[i + 1]) - 1) +
                            (( seq[j - 1]) - 1) )*6
                           + cp_shift];
    //Poly-C loop
    if( polyC == TRUE) {
      energy += em->polycslope*size + em->polycint;
    }
  }

  else if (size > 4) {
    // Calculate mismatch
    cp_shift = GetMismatchShift( seq[i], seq[j]);

    energy += em->MMEnergiesHP[(4*(( seq[i + 1]) - 1) +
                            (( seq[j - 1]) - 1) )*6
                           + cp_shift];

    //Poly-C loop
    if( polyC == TRUE) {
      energy += em->polycslope*size + em->polycint;
    }
  }
  return energy;
}



/* ****************************************** */
DEV
DBL_TYPE InteriorEnergy(  int i, int j, int h, int m, int seq[], energy_model_t
    * em) {
  return InteriorEnergyFull( i, j, h, m, seq, TRUE, em);
}

DEV
DBL_TYPE InteriorEnergyFull( int i, int j, int h, int m, int seq[],
                             int calcIJ, energy_model_t *em) {

  DBL_TYPE energy = 0.0;
  int L1, L2; //lengths of the 2 single stranded regions
  int size;
  int asymmetry;
  int cp_shift, ip_shift;  // For classifying basepairs

  if( em->dnarnacount == COUNT) return 0;
#ifdef DEBUG
  if( i >= h || h >= m || m >= j) {
    printf("Invalid boundary to interior loop! %d %d %d %d\n", i, h, m, j);
    assert(0);
  }
#endif

  L1 = h - i - 1;
  L2 = j - m - 1;
  size = L1 + L2;

  if( size == 0) { //Helical region
    energy = HelixEnergy( seq[i], seq[j], seq[h], seq[m], em);
  }

  else if ( L1*L2 == 0) { //Bulge
    if( size <= 30) {
      energy = em->loop37[ 30 + size - 1];
    }
    else {
      energy = em->loop37[ 30 + 30 - 1];
      energy += sizeLog (size, em); //1.75*kB*TEMP_K*LOG_FUNC( size/30.0);
    }

    if( L1 + L2 == 1 ) { //single bulge...treat as a stacked region
      energy += HelixEnergy( seq[i], seq[j], seq[h], seq[m], em);
      energy -= em->salt_correction;  // Correct for the extra salt correction
                                 // added from the HelixEnergy
    }
    else {
      // Next do AT_Penalty for no GC termination, assuming size >= 2
      if( seq[i] != BASE_C && seq[j] != BASE_C) {
        energy += em->at_penalty;
      }
      if( seq[h] != BASE_C && seq[m] != BASE_C) {
        energy += em->at_penalty;
      }
    }
  }
  else if ( L1 > 0 && L2 > 0) {
    asymmetry = abs( L1 - L2);
    if( asymmetry > 1 || size > 4) { //data not tabulated

      energy = asymmetryEfn( L1, L2, size, em);

      //Stacking Energy
      if( L1 > 1 && L2 > 1) { //Non-GAIL Version
        energy += InteriorMM( seq[m], seq[h], seq[m+1], seq[h-1], em);

        if( calcIJ == TRUE)
          energy += InteriorMM( seq[i], seq[j], seq[i+1], seq[j-1], em);
      }
      else if( L1 == 1 || L2 == 1) {// GAIL =>assume AA terminal mismatch
#ifndef NO_GAIL
        energy +=
          InteriorMM( seq[m], seq[h], BASE_A, BASE_A, em);
        if( calcIJ == TRUE)
          energy += InteriorMM( seq[i], seq[j], BASE_A, BASE_A, em);
#else
        energy += InteriorMM( seq[m], seq[h], seq[m+1], seq[h-1], em);
        if( calcIJ == TRUE)
          energy += InteriorMM( seq[i], seq[j], seq[i+1], seq[j-1], em)
#endif
      }
      else {
        printf("Error: Unclassified interior loop!\n");
        assert(0);
      }
    }
    else { //get tabulated data
      if( asymmetry == 0 && size == 2) {
        cp_shift = GetMismatchShift( seq[i], seq[j]);
        ip_shift = GetMismatchShift( seq[h], seq[m]);
        if (cp_shift==-1 || ip_shift==-1) return 0.0; //Wrongly called
        energy += em->IL_SInt2[ 96*cp_shift + 16*ip_shift +
                           4*(( seq[i+1]) - 1) +
                           (( seq[ j -1]) - 1) ];
      }
      else if( asymmetry == 0 && size == 4) {
        cp_shift = GetMismatchShift( seq[i], seq[j]);
        ip_shift = GetMismatchShift( seq[h], seq[m]);
        if (cp_shift==-1 || ip_shift==-1) return 0.0; //Wrongly called
        energy += em->IL_SInt4[ cp_shift*256*6 +  ip_shift*256 +
                           (4*(( seq[ i+1])  - 1) +
                            ( seq[ j - 1])   - 1)*16 +
                           (4*( ( seq[ i+2]) - 1) +
                            ( seq[ j - 2])   - 1) ];
      }
      else if( asymmetry == 1 && L1 == 1) {
        cp_shift = GetMismatchShift( seq[i], seq[j]);
        ip_shift = GetMismatchShift( seq[h], seq[m]);
        if (cp_shift==-1 || ip_shift==-1) return 0.0; //Wrongly called
        energy += em->IL_AsInt1x2[ cp_shift*4*24*4 +
                              (( seq[ j - 2]) - 1)*24*4 +
                              (( seq[ i + 1]) - 1)*24 +
                              4*ip_shift +
                              ((( seq[ j - 1]) - 1) % 4) ];
      }
      else if( asymmetry == 1 && L1 == 2) {
        cp_shift = GetMismatchShift( seq[j], seq[i]);
        ip_shift = GetMismatchShift( seq[m], seq[h]);
        if (cp_shift==-1 || ip_shift==-1) return 0.0; //Wrongly called
        //note reversed order of inputs above.
        //This is to comply with the format of asint1x2

        energy += em->IL_AsInt1x2[ ip_shift*4*24*4 +
                              (( seq[i + 1]) - 1)*24*4 +
                              (( seq[j - 1]) - 1)*24 +
                              4*cp_shift +
                              ((( seq[i + 2]) - 1) % 4) ];
      }
      else {
        printf("Error in tabulated Interior Loop!\n");
        assert(0);
      }
    }
  }
  else {
    printf("Improperly classified Interior Loop!\n");
    assert(0);
  }

  return energy;
}


/* ******************************** */
DEV
DBL_TYPE DangleEnergy( int i, int j, int seq[], int seqlength, energy_model_t
    *em) {
  //0 energy except for dangles

  DBL_TYPE dangle5 = 0;
  DBL_TYPE dangle3 = 0;
  int dangle_shift;

  if( em->dangletype != 2) {
    if( j == i - 1) {
      return 0;
    }
  }
  else if( j == i - 1 && (i == 0 || j == seqlength - 1) ) {
    return 0;
  }

  if( j == seqlength - 1) {
    dangle3 = 0;
  }
  else {
    int pt=GetPairType( seq[ j + 1]);
    if (pt==-1) {
      printf("i=%d j=%d seq[%d]=%d\n",i,j,j-1,seq[j-1]);
      assert(0);
    }
    dangle_shift = 3 - pt;
    dangle3 = em->dangle_energy[ 24 + dangle_shift*4 +
                            ( seq[ j]) - 1];
  }

  if( i == 0) {
    dangle5 = 0;
  }
  else {
    int pt=GetPairType( seq[ i - 1]);
    if (pt==-1) {
      printf("i=%d j=%d seq[%d]=%d\n",i,j,i-1,seq[i-1]);
      assert(0);
    }

    dangle_shift = pt;
    dangle5 = em->dangle_energy[ dangle_shift*4 +
                            ( seq[ i]) - 1];
  }

  if( em->dangletype != 2 && i == j && i != 0 && j != seqlength - 1) {
    return MIN(dangle3, dangle5 );
  }

  return dangle3 + dangle5;
}

/* ******************************** */
DEV
DBL_TYPE ExplDangleRaw( int i, int j, int seq[], int seqlength, energy_model_t
    *em) {
  //0 energy except for dangles

  DBL_TYPE dangle5 = 0;
  DBL_TYPE dangle3 = 0;
  int dangle_shift;

  if( (j == i - 1) || (j==-1 && i>0)) {
    return 1.0;
  }
  if( (j==-1 && i>0) || (j == i - 1 && (i == 0 || j == seqlength - 1)) ) {
    return 1.0;
  }

  if( j == seqlength - 1) {
    dangle3 = 0;
  }
  else {
    dangle_shift = 3 - GetPairType( seq[ j + 1]);
    dangle3 = em->dangle_energy[ 24 + dangle_shift*4 +
                            seq[ j] - 1];
  }

  if( i == 0) {
    dangle5 = 0;
  }
  else {
    dangle_shift = GetPairType( seq[i-1]);
    dangle5 = em->dangle_energy[ dangle_shift*4 +
                            seq[ i] - 1];
  }

  if(i == j && i != 0 && j != seqlength - 1) {
    if (em->dangletype == 2) return EXP_FUNC(-(dangle5 + dangle3)/(em->temp_k*kB));
    return EXP_FUNC( -MIN(dangle3, dangle5)/(em->temp_k*kB) );
  }

  return EXP_FUNC( -(dangle3 + dangle5)/(em->temp_k*kB) );
}

DEV
DBL_TYPE ExplDangle( int i, int j, int seq[], int seqlength, energy_model_t *em) {
  return ExplDangleRaw(i,j,seq,seqlength, em);
}


/* *********** */
DEV
DBL_TYPE NickDangle(int i, int j, const int *nicks, int **etaN, int hairpin,
                    int seq[], int seqlength, energy_model_t *em) {

  DBL_TYPE dangle5 = 0;
  DBL_TYPE dangle3 = 0;
  int dangle_shift;
  int nick;
  int nIndex;

  nick = -5;

  if( i != 0) { //if j == seqlength -1, this is still OK
    nIndex = EtaNIndex( i-0.5, j+0.5, seqlength);
  }
  else {
    nIndex = EtaNIndex( i+0.5, j+0.5, seqlength);
  }

  if( etaN[ nIndex][0] >= 2 ||
     ( etaN[ nIndex][0] == 1 && (i == 0 || j == seqlength -1)) ) {

       return NAD_INFINITY;
     }
  else if( etaN[ nIndex][0] >= 1) {
    nick = nicks[ etaN[ nIndex][1]];
  }

  if( em->dnarnacount == COUNT)
    return 0;

  if( j == i - 1) {
    return 0;
  }
  if( j == i - 1 && (i == 0 || j == seqlength - 1) ) {
    return 0;
  }

  if( j == seqlength - 1 || j == nick) {
    dangle3 = 0;
  }
  else {
    if( hairpin == FALSE) {
      dangle_shift = 3 - GetPairType( seq[ j + 1]);
    }
    else {
      dangle_shift = GetMismatchShift( seq[i-1], seq[j+1]);
    }

    dangle3 = em->dangle_energy[ 24 + dangle_shift*4 +
                            ( seq[ j]) - 1];
  }

  if( i == 0 || i-1 == nick) {
    dangle5 = 0;
  }
  else {
    if( hairpin == FALSE) {
      dangle_shift = GetPairType( seq[i-1]);
    }
    else {
      dangle_shift = GetMismatchShift( seq[i-1], seq[j+1]);
    }

    dangle5 = em->dangle_energy[ dangle_shift*4 +
                            ( seq[ i]) - 1];
  }

  if( nick >= i-1 && nick <= j) {
    return dangle3 + dangle5;
  }
  else {
    if( j > i || j == seqlength - 1 || i == 0) {
      return dangle3 + dangle5;
    }
    if(j == i && i != 0 && j != seqlength - 1) {
      if (em->dangletype == 2) return dangle3 + dangle5;
      return MIN(dangle3, dangle5);
    }
    //j == i-1 already handled above
  }
  printf("Error with function: NickDangle\n");
  assert(0);
  return -1; //Error!  This should never happen
}

/* ************** */
DEV
DBL_TYPE NickedEmptyQ( int i, int j, const int nicks[], int seq[],
                      int seqlength, int **etaN, energy_model_t *em) {

  if( j <= i || etaN[ EtaNIndex( i+0.5, j-0.5, seqlength)][0] == 0) {
    return EXP_FUNC( -1*NickDangle(i, j, nicks, etaN,
                              FALSE, seq, seqlength, em)
                /(kB*em->temp_k));
  }
  else { //disconnected
    return 0;
  }

}

/* ********* */
DEV
DBL_TYPE ExplInternal( int i, int j, int h, int m, int seq[], energy_model_t *em) {
  // Calculates E^(-energy/RT) of interior loop closed by i-j and h-m

  DBL_TYPE energy = InteriorEnergy( i, j, h, m, seq, em);
  if( energy == NAD_INFINITY) {
    return 0.0;
  }
  return EXP_FUNC( - energy/( kB*em->temp_k));
}

DEV
DBL_TYPE sizeLog(int size, energy_model_t *em){
  return 1.75*kB*em->temp_k*LOG_FUNC(size/30.0);
}


/* ******* */
DEV
DBL_TYPE asymmetryEfn( int L1, int L2, int size, energy_model_t *em) {

  int asymmetry_index;
  DBL_TYPE energy;
  int asymmetry = abs( L1 - L2);

  //Loop Size Energy
  if( size <= 30) {
    energy = em->loop37[ size - 1];
  }
  else {
    energy = em->loop37[ 30 - 1];
    energy += sizeLog(size, em);
  }

  //Asymmetry rountine copied from efn.f in Zuker's mfold package.
  asymmetry_index = 4;

  if( L1 < asymmetry_index) {
    asymmetry_index = L1;
  }

  if( L2 < asymmetry_index) {
    asymmetry_index = L2;
  }

  if( asymmetry*em->asymmetry_penalty[ asymmetry_index - 1] < em->max_asymmetry ) {
    energy += asymmetry*em->asymmetry_penalty[ asymmetry_index - 1];
  }
  else {
    energy += em->max_asymmetry; // MAX asymmetry penalty
  }
  return energy;
}



/* ********************** */
