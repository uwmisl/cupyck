#include "hip/hip_runtime.h"
/*
  pf.c is part of the NUPACK software suite
  Copyright (c) 2007 Caltech. All rights reserved.
  Coded by: Robert Dirks 3/2006, Justin Bois 1/2007
            Asif Khan 8/2009 Brian Wolfe 10/2009
  This file moves the partition function algorithm to a function, so
  that it can be more readily used as in a library.  In addition,
  scaling is incorporated to allow for longer sequences.
*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>

#include "pfuncUtilsHeader.h" //contains functions and structures
#include "DNAExternals.h"


/* ************************************************ */
// This is the main function for computing partition functions.

DEV
int getNodesFirstEntry(int L, int rank, int seqLen, int N) {
    int split, rem, last;
      last = seqLen - L + 1;
        if (rank >= N || rank < 0)
              return last; 
          split = (int)(last/N);
            rem = last % N;
              return rank*split + MIN(rank, rem);
}

DEV
int IsProcessorInUse(int rank, int L, int seqLen, int numCPUs ) {
    return (getNodesFirstEntry(L,rank,seqLen,numCPUs) != seqLen - L + 1);
}

__constant__ energy_model_t *energies;

GLB
void pfuncFullWithSymHelper(DBL_TYPE *pf, int ** inputSeqs, int * seqlengths,
    int * nStrands_arr, int * permSymmetries) {

  int * inputSeq = inputSeqs[blockIdx.x];
  int seqlength = seqlengths[blockIdx.x];
  int nStrands = nStrands_arr[blockIdx.x];
  int permSymmetry = permSymmetries[blockIdx.x];
  //complexity: 3 = N^3, 4 = N^4, 5 = N^5, 8 = N^8
  //naType: DNA = 0, RNA = 1
  //dangles: 0 = none, 1 = normal, 2 = add both

  __shared__ int *seq;
  if(threadIdx.x == 0) {
    seq = (int*)malloc((seqlength + 1) * sizeof(int));
  }

  __shared__ DBL_TYPE *Q;
  __shared__ DBL_TYPE *Qb;
  __shared__ DBL_TYPE *Qm; //O(N^2)


  //N^3 arrays
  __shared__ DBL_TYPE *Qx;
  __shared__ DBL_TYPE *Qx_1;
  __shared__ DBL_TYPE *Qx_2;
  __shared__ DBL_TYPE *Qs;
  __shared__ DBL_TYPE *Qms;

  /*
  The above matrices are dynamically allocated matrices that
  contain partition functions restricted to a subsequence of the
  strand.  Each of the above should be accessed by the call
  to Q[ pf_index(i, j)] indicate the partition function between
  i and j, inclusive. 

  They are described in the paper mentioned above.
  */

  int i, j; // the beginning and end bases for Q;
  int L; //This the length of the current subsequence 
  int pf_ij; //index for O(N^2) matrixes; used to reduce calls to pf_index;
  DBL_TYPE returnValue;


  int iMin;
  int iMax;
  

  __shared__ int nicks[ MAXSTRANDS];  //the entries must be strictly increasing
  if (threadIdx.x == 0) {
    for (i=0;i<MAXSTRANDS;i++){
      nicks[i]=-1;
    }
  }
  //nicks[i] = N means a strand ends with base N, and a new one starts at N+1
  // isNicked[n] is 0 if no nick at n, 1 otherwise

  __shared__ int **etaN;
  __shared__ int *etaN_space;

  if (threadIdx.x == 0) {
    processMultiSequence( inputSeq, seqlength, nStrands, seq, nicks);

    // Allocate and Initialize Matrices
    int arraySize = seqlength*(seqlength+1)/2+(seqlength+1);

    InitLDoublesMatrix( &Q, arraySize, "Q");
    InitLDoublesMatrix( &Qb, arraySize, "Qb");
    InitLDoublesMatrix( &Qm, arraySize, "Qm");

    etaN = (int**)malloc(arraySize * sizeof(int*));
    etaN_space = (int*)malloc(arraySize * 2 * sizeof(int));
    for (int i = 0; i < arraySize; ++i) {
      etaN[i] = etaN_space + (2 * i);
    }
    InitEtaN( etaN, nicks, seqlength);
    nonZeroInit( Q, seq, seqlength, energies);

    InitLDoublesMatrix( &Qs, arraySize, "Qs");
    InitLDoublesMatrix( &Qms, arraySize, "Qms");

    InitLDoublesMatrix( &Qx, arraySize/2, "Qx");
    InitLDoublesMatrix( &Qx_1, arraySize/2, "Qx_1");
    InitLDoublesMatrix( &Qx_2, arraySize/2, "Qx_2");

  }

  for( L = 1; L <= seqlength; L++) {
    /* Calculate all sub partition functions for
    distance = 0, then 1, then 2.... */

    if (threadIdx.x == 0) {
      manageQx( &Qx, &Qx_1, &Qx_2, L-1, seqlength);   
    }
    __syncthreads();

    iMin = getNodesFirstEntry(L, threadIdx.x, seqlength, blockDim.x);
    iMax = getNodesFirstEntry(L, threadIdx.x + 1, seqlength, blockDim.x) - 1;
    /*
    //Default without parallelization
    iMin = 0;
    iMax = seqlength - L; 
    */

  
    int active = IsProcessorInUse(threadIdx.x, L, seqlength, blockDim.x);
    for( i = iMin; i <= iMax && active; i++) {
      j = i + L - 1;
      pf_ij = pf_index( i, j, seqlength);
      /* Recursions for Qb.  See figure 13 of paper */
      /* bp = base pairs, pk = pseudoknots */
      if( CanPair( seq[ i], seq[ j]) == FALSE) {
        Qb[ pf_ij] = 0.0; //scaling still gives 0
      }
      else {
        Qb[ pf_ij] = ExplHairpin( i, j, seq, seqlength, etaN, energies);

        //no nicked haripins allowed in previous function
        if( etaN[ EtaNIndex(i+0.5, i+0.5, seqlength)][0] == 0 &&
           etaN[ EtaNIndex(j-0.5, j-0.5, seqlength)][0] == 0) {
             //regular multiloop.  No top-level nicks
             
              Qb[ pf_ij] += SumExpMultiloops(i, j, seq, Qms, Qm,
                                            seqlength, etaN, energies);
        }

        if( etaN[ EtaNIndex(i+0.5, j-0.5, seqlength)][0] >= 1) {
          //Exterior loop (created by nick)
          Qb[ pf_ij] += SumExpExteriorLoop( i, j, seq, seqlength, 
                                           Q, nicks, etaN, energies); 
        }

      }

      fastILoops( i, j, L, seqlength, seq, etaN, Qb, Qx, Qx_2, energies);


      /* Recursions for Qms, Qs */
      MakeQs_Qms( i, j, seq, seqlength, Qs, Qms, Qb, nicks, etaN, energies);
      
      /* Recursions for Q, Qm, Qz */
      MakeQ_Qm_N3( i, j, seq, seqlength, Q, Qs, Qms, Qm,
                  nicks,etaN, energies);


    }
    __syncthreads();
  }

  //adjust this for nStrands, symmetry at rank == 0 node
  if (threadIdx.x == 0) {
    returnValue = EXP_FUNC(
        -1*(energies->bimolecular + energies->salt_correction)*(nStrands-1)/
        (kB*energies->temp_k)
      ) * Q[ pf_index(0,seqlength-1, seqlength)]/((DBL_TYPE) permSymmetry);

    free( Q);
    free( Qb);
    free( Qm);

    Q = Qb = Qm = NULL;

    free( Qs);
    free( Qms);
    
    free( Qx);
    free( Qx_1);
    free( Qx_2);
    
    Qs = Qms = Qx = Qx_1 = Qx_2 = NULL;

    free( seq);

    for( i = 0; i <= seqlength-1; i++) {
      for( j = i-1; j <= seqlength-1; j++) {
        pf_ij = pf_index(i,j,seqlength);
        free( etaN[pf_ij]);
      }
    }

    free( etaN);
    pf[blockIdx.x] = returnValue;
  }
}
/* ****** */
//void pfuncInitialize(DBL_TYPE temp_lo, DBL_TYPE temp_hi, DBL_TYPE temp_step,

DBL_TYPE * pf;
int ** intSeqs;
int * seqlengths;
int * nStrands_arr;
int * permSymmetries;
int nblocks;

#define cudaCheck(call) {\
  hipError_t e = (call); \
  if(e != hipSuccess) {\
    fprintf(stderr, #call ": %s", hipGetErrorString(e)); \
    abort(); \
  }\
}\


extern "C" void pfuncInitialize(int nblocks_in, DBL_TYPE temp_k,
    DBL_TYPE sodium_conc, DBL_TYPE magnesium_conc,
    int long_helix, int dangletype, int dnarnacount) {

  nblocks = nblocks_in;
  SODIUM_CONC = sodium_conc;
  MAGNESIUM_CONC = magnesium_conc;
  USE_LONG_HELIX_FOR_SALT_CORRECTION = long_helix;
  DANGLETYPE = dangletype;
  DNARNACOUNT = dnarnacount;

  // Load Energy Model
  energy_model_t em;
  LoadEnergies(&em, temp_k);

  // Allocate Device Memory
  energy_model_t *em_d;
  hipMalloc(&em_d, sizeof(energy_model_t));
  hipMemcpy(em_d, &em, sizeof(energy_model_t), hipMemcpyHostToDevice);

  // Send Device Ptr to Device
  hipMemcpyToSymbol(HIP_SYMBOL(energies), &em_d, sizeof(energy_model_t*));

  //hipMemcpyToSymbol(HIP_SYMBOL(ENERGIES), &energies, sizeof(energy_model_t));

  cudaCheck(
    hipMallocManaged(&pf, nblocks * sizeof(DBL_TYPE))
  );
  cudaCheck(
    hipMallocManaged(&seqlengths, nblocks * sizeof(int))
  );
  cudaCheck(
    hipMallocManaged(&nStrands_arr, nblocks * sizeof(int))
  );
  cudaCheck(
    hipMallocManaged(&permSymmetries, nblocks * sizeof(int))
  );
  cudaCheck(
    hipMallocManaged(&intSeqs, nblocks * sizeof(int*))
  );

  for(int i = 0; i < nblocks; ++i) {
    cudaCheck(
      hipMallocManaged(&(intSeqs[i]), (MAXSEQLENGTH + 1) * sizeof(int))
    );
  }
  cudaCheck(
    hipDeviceSetLimit(hipLimitMallocHeapSize, 1 << 30)
  );
}

extern "C" void pfuncMulti(char ** inputSeqs, int nseqs, int * permSym, DBL_TYPE
    * result) {

  for (int s = 0; s < nseqs; s += nblocks) {
    for(int i = 0; i < nblocks; ++i) {
      int len = strlen(inputSeqs[s + i]);
      convertSeq(inputSeqs[s + i], intSeqs[i], len);
      seqlengths[i] = getSequenceLengthInt(intSeqs[i], nStrands_arr + i);
      permSymmetries[i] = permSym[s + i];
    }

    pfuncFullWithSymHelper<<<nblocks, 256>>>(
        pf, intSeqs, seqlengths, nStrands_arr, permSymmetries
    );
    hipDeviceSynchronize();

    for(int i = 0; i < nblocks; ++i) {
      result[s + i] = pf[i];
    }
  }
}
