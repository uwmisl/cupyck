/*  
    pfunc.c is part of the NUPACK software suite
    Copyright (c) 2007 Caltech. All rights reserved.
    Coded by: Robert Dirks, 5/2005 and Justin Bois 1/2007   
    
    The purpose of this program is to calculate the partition function
    of all possible secondary structures of a given strand (or
    strands) of DNA/RNA.
    
    If there are multiple strands, then this algorithm will calculate
    the partition function for a single circular permutation of those
    strands, assuming each strand to be distinguishable from the
    others.  This algorithm is described in our SIAM Review paper
    published in 2007 (Dirks, Bois, Schaeffer, Winfree, Pierce).  The
    time complexity of this algorithm is O(N^3), where N is the total
    sequence of all the strands involved.
    
    For a single strand, the algorithm can be expanded to allow for
    the simplest kinds of pseudoknots, but with computational
    complexity of O(N^5), and storage complexity of O(N^4).  This
    algorithm is described in (Dirks, Pierce JCC 24:1664-77, 2003),
    and (Dirks, Pierce, JCC 25:1295-1304, 2004).  The Qp portion of
    the algorithm has been recently expanded from the paper to allow
    for gap spanning regions containing a single base pair.
    
    pfunc.c will compile as a stand alone executable to calculate
    partition functions.  The algorithms can also be compiled as a
    static library to be used by other code.
    
    Default pfunc.c usage: The default setings include the mfold2.3
    RNA parameter set at 37C.  Pseudoknots are enabled with the
    -pseudo flag unless the input sequence has multiple strands.
    These default parameters can be changed by modifying the call to
    pfuncFull.
*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <string.h>
#include <signal.h>


#include "pfuncUtilsHeader.h"
#include "DNAExternals.h"

int readMultiInput(char *inputFile, char ***seqs) {
  FILE *fin = fopen(inputFile, "r");
  if (!fin) {
    fprintf(stderr, "unable to open file\n");
    exit(1);
  }
  int nlines = 0;
  int rc = 0;
  while (rc != EOF) {
    rc = fgetc(fin);
    if (rc == '\n') {
      ++nlines;
    }
  }

  *seqs = (char**)malloc(nlines * sizeof(char*));
  rewind(fin);

  size_t len = MAXSEQLENGTH;
  for(int i = 0; i < nlines; ++i) {
    (*seqs)[i] = (char*)malloc(MAXSEQLENGTH);
    int nc = getline(&((*seqs)[i]), &len, fin);
    if (len > MAXSEQLENGTH) {
      fprintf(stderr, "seqs longer than MAXSEQLENGTH");
      exit(1);
    }
    (*seqs)[i][nc-1] = '\0';
  }
  fclose(fin);
  return nlines;
}

/* ************************************************ */

int main( int argc, char *argv[] ) {
  
  char seq[ MAXSEQLENGTH];
  int seqNum[ MAXSEQLENGTH+1];
  
  DBL_TYPE pf;

  int vs;
  int tmpLength;
  char inputFile[ MAXLINE];
  int inputFileSpecified;
  
  strcpy( inputFile, "");
  
  inputFileSpecified = ReadCommandLineNPK( argc, argv, inputFile);
  
  energy_model_t energy_model;
  energy_model.temp_k = TEMP_K;
  energy_model.dangletype = DANGLETYPE;
  energy_model.dnarnacount = DNARNACOUNT;
  LoadEnergies(&energy_model, TEMP_K);

  hipMemcpyToSymbol(HIP_SYMBOL(ENERGIES), &energy_model, sizeof(energy_model_t));

  if(NupackShowHelp) {
    printf("Usage: pfunc [OPTIONS] PREFIX\n");
    printf("Calculate the partition function of the input sequence.\n");
    printf("Example: pfunc -multi -T 25 -material dna example\n");
    PrintNupackThermoHelp();
    PrintNupackUtilitiesHelp();
    exit(1);
  }

  header( argc, argv, "pfunc","screen");
  if( !inputFileSpecified || 
      !ReadInputFile( inputFile, seq, &vs, NULL, NULL, NULL) ) {
       if (inputFileSpecified == 0) getUserInput( seq, &vs, NULL, NULL);
       else abort();
  }
  
  printInputs( argc, argv, seq, vs, NULL, NULL,"screen");
  
  //calculate partition function, without pairs info
  tmpLength = strlen( seq);
  convertSeq(seq, seqNum, tmpLength);

  pf = pfuncFullWithSym(seqNum, vs);

  printf("%s\n%s Free energy (kcal/mol) and partition function:\n",
	 COMMENT_STRING,COMMENT_STRING);

  printf("%.8Le\n",-1*(kB*TEMP_K)*logl( (long double) pf));
  printf( "%12.14Le\n", (long double) pf); 
  
  return 0;
}
/* ****** */













